#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"

extern "C" {

#include "ddi.h"

extern double MPI_Wtime();

#define CUDA_RC_CHECK(a) \
        do { a; \
        hipError_t err = hipGetLastError();                                                      \
        if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
           printf(" +----------------------------------------\n"); \
           printf(" | ** CUDA ERROR! ** \n"); \
           printf(" | Error: \n"); \
           printf(" | Msg: %s\n", hipGetErrorString(err) ); \
           printf(" | File: %s\n", __FILE__ );  \
           printf(" | Line: %d\n", __LINE__ );  \
           printf(" +----------------------------------------\n"); \
                 exit(-1);  \
           } \
        } while(0)

#define CUBLAS_RC_CHECK(a) do { a; } while(0)

#define SD_SHIFT(ptr,shift) \
        if(ptr == ptr ## _tail) ptr = ptr ## _head; \
        else                    ptr += shift;

#define CUDA_EVENT_COUNT        2
#define CUDA_D2H_EVENT_COUNT    3

static  hipblasStatus_t stat;
static  hipblasHandle_t cublas_hnd;
static  hipError_t cudaStat;

typedef struct {
        int * ddi_handle;
} sd_t;

void streaming_dgemm(int m, int n, int k, double alpha, const sd_t *a, const sd_t *b,
                     double beta, const sd_t *c, int tm, int tn, int tk);

int main(int argc, char *argv[])
{
        int me, np, my, nn;
        int ddi_a, ddi_b, ddi_c;

     // read in m, n, k
        int m = atoi(argv[1]);
        int n = atoi(argv[2]);
        int k = atoi(argv[3]);

     // read in tile dimensions
        int tm = atoi(argv[4]); 
        int tn = atoi(argv[5]);
        int tk = atoi(argv[6]);

     // determine distriubted matrix requirements
        size_t dm = ( (long)m*k + (long)k*n + (long)m*n ) * 1.5;
        dm /= 1000000;

     // initialized ddi / mpi
        DDI_Init(argc,argv);
        DDI_NProc(&np, &me);
        DDI_NNode(&nn, &my);
        DDI_Memory(dm);

     // Create Full A, B and C
        DDI_Create(m, k, &ddi_a);
        DDI_Create(k, n, &ddi_b);
        DDI_Create(m, n, &ddi_c);

        sd_t a, b, c;
        a.ddi_handle = &ddi_a;
        b.ddi_handle = &ddi_b;
        c.ddi_handle = &ddi_c;

     // load cublas
        stat = hipblasCreate( &cublas_hnd );

     // call streaming dgemm
        streaming_dgemm( m, n, k, 1.0, &a, &b, 1.0, &c, tm, tn, tk );

     // Clean up memory - distributed
        DDI_Destroy( ddi_c );
        DDI_Destroy( ddi_b );
        DDI_Destroy( ddi_a );

     // Destory cublas handle
        stat = hipblasDestroy( cublas_hnd );

     // Finalize
        DDI_Finalize();
        return 0;
}

void streaming_dgemm(int m, int n, int k, double alpha, const sd_t *a, const sd_t *b, 
                     double beta, const sd_t *c, int tm, int tn, int tk)
{
     // id
        int me, np, my, nn;

     // timing
        double start_time, stop_time;

     // ddi handles
        int ddi_a = *(a->ddi_handle);
        int ddi_b = *(b->ddi_handle);
        int ddi_c = *(c->ddi_handle);

     // id
        DDI_NProc(&np, &me);
        DDI_NNode(&nn, &my);

     // determine the number of patches
        int m_patch_count = (m + tm - 1) / tm;
        int n_patch_count = (n + tn - 1) / tn;
        int k_patch_count = (k + tk - 1) / tk;
        int tiled_dgemm_count = m_patch_count * n_patch_count * k_patch_count;

     // Host Memory - define
        size_t ta_count = (long)tm * (long)tk;
        size_t tb_count = (long)tk * (long)tn;
        size_t tc_count = (long)tm * (long)tn;

        size_t ta_size = ta_count * sizeof(double);
        size_t tb_size = tb_count * sizeof(double);
        size_t tc_size = tc_count * sizeof(double);

        size_t h_a_size = ta_size * 3;
        size_t h_b_size = tb_size * 3;
        size_t h_c_size = tc_size * 3;

        double h_a_size_in_mb = (double) h_a_size / (1024*1024);
        double h_b_size_in_mb = (double) h_b_size / (1024*1024);
        double h_c_size_in_mb = (double) h_c_size / (1024*1024);
        double h_size_in_mb = h_a_size_in_mb + h_b_size_in_mb + h_c_size_in_mb;

     // Host Memory = alloc
        double * h_a = (double *) malloc( h_a_size );
        double * h_b = (double *) malloc( h_b_size );
        double * h_c = (double *) malloc( h_c_size );

     // Host Memory - register
        CUDA_RC_CHECK( hipHostRegister( h_a, h_a_size, 0 ) );
        CUDA_RC_CHECK( hipHostRegister( h_b, h_b_size, 0 ) );
        CUDA_RC_CHECK( hipHostRegister( h_c, h_c_size, 0 ) );

     // Device Memory - define
        size_t d_a_size = ta_count * sizeof(double) * 2;
        size_t d_b_size = tb_count * sizeof(double) * 2;
        size_t d_c_size = tc_count * sizeof(double) * 2;

        double d_a_size_in_mb = (double) d_a_size / (1024*1024);
        double d_b_size_in_mb = (double) d_b_size / (1024*1024);
        double d_c_size_in_mb = (double) d_c_size / (1024*1024);
        double d_size_in_mb = d_a_size_in_mb + d_b_size_in_mb + d_c_size_in_mb;

     // Device Memory - alloc
        double * d_a = NULL;
        double * d_b = NULL;
        double * d_c = NULL;
        CUDA_RC_CHECK( hipMalloc( (void **) &d_a, d_a_size ) );
        CUDA_RC_CHECK( hipMalloc( (void **) &d_b, d_b_size ) );
        CUDA_RC_CHECK( hipMalloc( (void **) &d_c, d_c_size ) );
        CUDA_RC_CHECK( hipMemset( d_c, 0, d_c_size ) );

     // Create CUDA Streams
        hipStream_t * stream = (hipStream_t *) malloc( sizeof(hipStream_t) * CUDA_STREAM_COUNT );

     // Initialize Streams & Record Initial events
        for(int i=0; i<CUDA_STREAM_COUNT; i++)  CUDA_RC_CHECK( hipStreamCreate( &stream[i] ) );

     // Create CUDA Events
        hipEvent_t * event     = (hipEvent_t *) malloc( sizeof(hipEvent_t) * CUDA_EVENT_COUNT );
        hipEvent_t * d2h_event = (hipEvent_t *) malloc( sizeof(hipEvent_t) * CUDA_D2H_EVENT_COUNT );
        
     // Initialzie Cuda Events
        for(int i=0; i<CUDA_EVENT_COUNT; i++)     CUDA_RC_CHECK( hipEventCreate( &event[i] ) );
        for(int i=0; i<CUDA_D2H_EVENT_COUNT; i++) CUDA_RC_CHECK( hipEventCreate( &d2h_event[i] ) );
     

     // Prepare buffering pointers
        double * h_a_head = h_a;
        double * h_a_tail = h_a + ta_count*2;
        double * h_b_head = h_b;
        double * h_b_tail = h_b + tb_count*2;
        double * h_c_head = h_c;
        double * h_c_tail = h_c + tc_count*2;

        double * d_a_head = d_a;
        double * d_a_tail = d_a + ta_count;
        double * d_b_head = d_b;
        double * d_b_tail = d_b + tb_count;
        double * d_c_head = d_c;
        double * d_c_tail = d_c + tc_count;

        hipStream_t * stream_head = stream;
        hipStream_t * stream_tail = stream + 1;

        if(me == 0) 
        {
           printf("\n");
           printf("---------------- Streaming DGEMM using DDI ---------------- \n"); 
           printf("global dimensions: %8d %8d %8d\n",m,n,k);
           printf("tile dimensions  : %8d %8d %8d\n",tm,tn,tk);
           printf("\n");
           printf("dividing %d work packets over %d nodes\n", (m_patch_count*n_patch_count),nn);
           printf("each work packet consists of %d tiled dgemms\n",k_patch_count);
           printf("\n");
           printf("host requirements   = %.2lf + %.2lf + %.2lf = %.2lf\n", h_a_size_in_mb,
                   h_b_size_in_mb, h_c_size_in_mb, h_size_in_mb);
           printf("device requirements = %.2lf + %.2lf + %.2lf = %.2lf\n", d_a_size_in_mb,
                   d_b_size_in_mb, d_c_size_in_mb, d_size_in_mb);
           printf("\n");
           fflush(stdout);
        }

        DDI_Sync(1234);

     // DDOT Algorithm - Start Up
     // Dynamically load-balance all incoming patches
     // Stream patches of A & B from distributed memory 
     // C remains device resident over the summing index

        size_t patch_count = m_patch_count * n_patch_count;
        DDI_Patch a_patch, b_patch;
        DDI_Patch * c_patch = (DDI_Patch *) malloc( sizeof(DDI_Patch) * H_C_PATCH_COUNT );
        DDI_Patch * c_patch_head = c_patch;
        DDI_Patch * c_patch_tail = c_patch + H_C_PATCH_COUNT - 1;
        DDI_Patch * put_patch = c_patch;
        DDI_Patch * put_patch_head = c_patch_head;
        DDI_Patch * put_patch_tail = c_patch_tail;
        
        int * remaining_iterations = (int *) malloc( sizeof(int) * H_C_PATCH_COUNT );
        int * remaining_iterations_head = remaining_iterations;
        int * remaining_iterations_tail = remaining_iterations + H_C_PATCH_COUNT - 1; 

        double * hc_put   = hc;

        long ip = 0;            // number of patches streamed from the network
        size_t dlb_counter;     // dynamic load balancer

        double _one  = 1.0;
        double _zero = 0.0;
        const double *one = &_one;
        const double *zero = &_zero;

        double *palpha = &alpha;
        double *pbeta  = &beta;

        DDI_DLBNext(&dlb_counter);

        start_time = MPI_Wtime();
        while(dlb_counter < patch_count)
        {
           int patch_coord_i = dlb_counter / m_patch_count;
           int patch_coord_j = dlb_counter % m_patch_count;
        // Constant over these patch coordinates (i,j)
           a_patch.ilo = tm * patch_coord_i;
           a_patch.ihi = a_patch.ilo + tm - 1;
           b_patch.jlo = tn * patch_coord_j;
           b_patch.jhi = b_patch.jlo + tn - 1;
           for(int ik=0; ik<k_patch_count; ik++,ip++)
           {
               if( ip > k_patch_count ) {
                   if( *remaining_iterations == 1) CUDA_RC_CHECK( hipEventSynchronize( d2h_event ) ); // it has to be done this time
                   cudaQuery = hipEventQuery( d2h_event );  // does this return true every - i think yes (it is a value < the current event counter val)
                   if( *remaining_iterations == 1) assert( cudaQuery == hipSuccess );
                   if(cudaQuery != hipSuccess) break;
                   ncols_remaining = put_patch->jhi - put_patch.jlo + 1; 
                   if(*remaining_iterations > k_patch_count) *remaining_iterations = k_patch_count; // this allows use to finish early!
                   cols_per_iteration = ncols_remaining / *remaining_iterations; // if remaining_iterations is > k_patch_count, then we are ahead of 
                   memcpy(patch, put_patch, sizeof(DDI_Patch));                  // schedule and we are try to finish ahead of schedule
                   patch.jhi = patch.jlo + cols_per_iteration;
                   put_patch.jlo = patch.jhi + 1;
                   DDI_Put(ddi_c, &patch, hc_put);
                   hc_put += patch->size; // ensure ddi sets size
                   (*remaining_iterations)--;
                   if(*remaining_iterations == 0) {
                      SD_SHIFT( put_patch, 1 );
                      SD_SHIFT( d2h_event, 1 );
                      SD_SHIFT( h_c, tc_count );
                      SD_SHIFT( remaining_iterations, 1 );
                      hc_put = h_c;
               }   }
               a_patch.jlo = tk * ik;
               a_patch.jhi = a_patch.jlo + tk - 1;
               b_patch.ilo = a_patch.jlo;
               b_patch.ihi = a_patch.jhi;
               DDI_GetP(ddi_a, &a_patch, h_a);
               DDI_GetP(ddi_b, &b_patch, h_b);
               if(ip > 1) CUDA_RC_CHECK( hipEventSynchronize( *event ) );
               CUDA_RC_CHECK( hipMemcpyAsync( d_a, h_a, ta_size, hipMemcpyHostToDevice, *stream ) );
               CUDA_RC_CHECK( hipMemcpyAsync( d_b, h_b, tb_size, hipMemcpyHostToDevice, *stream ) );
               CUBLAS_RC_CHECK( hipblasSetStream( cublas_hnd, *stream ) );
               CUBLAS_RC_CHECK( hipblasDgemm( cublas_hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                             tn, tm, tk, palpha, d_a, tn, d_b, tk, pbeta, d_c, tn ) );
               CUDA_RC_CHECK( hipEventRecord( *event, stream ) );
               if(ik == patch_count-1) {
                  CUDA_RC_CHECK( hipMemcpyAsync( h_c, d_c, tc_size, hipMemcpyDeviceToHost, *stream ) );
                  CUDA_RC_CHECK( hipEventRecord( d2h_event, stream ) );
                  *remaining_iterations = 2 * k_patch_count;
                  SD_SHIFT( d_c, tc_count );
               }
               SD_SHIFT( h_a, ta_count ); 
               SD_SHIFT( h_b, tb_count );
               SD_SHIFT( d_a, ta_count );
               SD_SHIFT( d_b, tb_count );
               SD_SHIFT( stream , 1 );
               SD_SHIFT( event , 1 );
           } // end ik loop over k_patch_count
           c_patch->ilo = a_patch.ilo;
           c_patch->ihi = a_patch.ihi;
           c_patch->jlo = b_patch.jlo;
           c_patch->jhi = b_patch.jhi;
           SD_SHIFT( c_patch, 1 );
           DDI_DLBNext(&dlb_counter);
        } // end while loop

        // the only thing we need to do outside the loops is process putting hc back into ddi_c!

        // finished loops ==> drain buffers - step 1 of 2
        // moves last get on to the device & executes the dgemm for the data already there
        if(ip > 1)
        {
            CUBLAS_RC_CHECK( hipblasSetStream( cublas_hnd, *stream ) );
            CUBLAS_RC_CHECK( hipblasDgemm( cublas_hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                          tn, tm, tk, palpha, d_a, tn, d_b, tk, pbeta, d_c, tn ) );
            SD_SHIFT( stream, 1 );
            SD_SHIFT( d_a, ta_count );
            SD_SHIFT( d_b, tb_count );
        }
  
        hipStreamSynchronize( *stream );
        // add copy c off logic in here
        CUDA_RC_CHECK( hipMemcpyAsync( d_a, h_a, ta_size, hipMemcpyHostToDevice, *stream ) );
        CUDA_RC_CHECK( hipMemcpyAsync( d_b, h_b, tb_size, hipMemcpyHostToDevice, *stream ) );
        SD_SHIFT( h_a, ta_count );
        SD_SHIFT( h_b, tb_count );

        ++ip;

        // finished loops ==> drain buffers - step 2 of 2
        CUBLAS_RC_CHECK( hipblasSetStream( cublas_hnd, *stream ) );
        CUBLAS_RC_CHECK( hipblasDgemm( cublas_hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                      tn, tm, tk, palpha, d_a, tn, d_b, tk, pbeta, d_c, tn ) );

        // move last patch of c off the device
        CUDA_RC_CHECK( hipMemcpyAsync( h_c, d_c, tc_size, hipMemcpyDeviceToHost, *stream ) );
        CUDA_RC_CHECK( hipStreamSynchronize( *stream ) );

        DDI_Sync(1234);
        stop_time = MPI_Wtime();

        if(me == 0) {
           printf("walltime = %.6lf\n", (stop_time - start_time));
           printf("\n");
           printf("---------------- Streaming DGEMM using DDI ---------------- \n"); 
           fflush(stdout);
        }

     // Destroy Streams
        CUDA_RC_CHECK( hipStreamSynchronize( c_stream ) );
        CUDA_RC_CHECK( hipStreamDestroy( c_stream ) );
        // for(int i=0; i<2; i++) {
        //    CUDA_RC_CHECK( hipStreamSynchronize( stream[i] ) );
        //    CUDA_RC_CHECK( hipStreamDestroy( stream[i] ) );
       //  }

     // Clean up memory - host
        free( h_a_head );
        free( h_b_head );
        free( h_c_head );

     // Clean up memory - device
        CUDA_RC_CHECK( hipFree( d_a_head ) );
        CUDA_RC_CHECK( hipFree( d_b_head ) );
        CUDA_RC_CHECK( hipFree( d_c_head ) );
}
}

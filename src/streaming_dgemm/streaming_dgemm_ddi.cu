#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"

extern "C" {

#include "ddi.h"

extern double MPI_Wtime();

#define CUDA_RC_CHECK(a) \
        do { a; \
        hipError_t err = hipGetLastError();                                                      \
        if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
           printf(" +----------------------------------------\n"); \
           printf(" | ** CUDA ERROR! ** \n"); \
           printf(" | Error: \n"); \
           printf(" | Msg: %s\n", hipGetErrorString(err) ); \
           printf(" | File: %s\n", __FILE__ );  \
           printf(" | Line: %d\n", __LINE__ );  \
           printf(" +----------------------------------------\n"); \
                 exit(-1);  \
           } \
        } while(0)

#define CUBLAS_RC_CHECK(a) do { a; } while(0)

int main(int argc, char *argv[])
{
        int me, np, my, nn;
        int ddi_a, ddi_b, ddi_c;
        double start_time, stop_time;
        hipblasStatus_t stat;
        hipblasHandle_t cublas_hnd;
        hipError_t cudaStat;

     // load cublas
        stat = hipblasCreate( &cublas_hnd );

     // read in m, n, k
        printf("argc=%d\n",argc);
        fflush(stdout);
        int m = atoi(argv[1]);
        int n = atoi(argv[2]);
        int k = atoi(argv[3]);

     // read in tile dimensions
        int tm = atoi(argv[4]);
        int tn = atoi(argv[5]);
        int tk = atoi(argv[6]);

     // determine the number of patches
        int m_patch_count = (m + tm - 1) / tm;
        int n_patch_count = (n + tn - 1) / tn;
        int k_patch_count = (k + tk - 1) / tk;
        int tiled_dgemm_count = m_patch_count * n_patch_count * k_patch_count;

     // determine distriubted matrix requirements
        printf("m,n,k = %d, %d, %d\n",m,n,k);
        fflush(stdout);
        size_t dm = ( (long)m*k + (long)k*n + (long)m*n ) * 3;
        printf("dm = %ld\n",dm);
        fflush(stdout);
        dm /= 1000000;
        printf("dm = %ld\n",dm);
        fflush(stdout);

     // initialized ddi / mpi
        DDI_Init(argc,argv);
        DDI_NProc(&np, &me);
        DDI_NNode(&nn, &my);
        DDI_Memory(dm);

     // Create Full A, B and C
        DDI_Create(m, k, &ddi_a);
        DDI_Create(k, n, &ddi_b);
        DDI_Create(m, n, &ddi_c);

     // Host Memory - define
        size_t ta_count = tm * tk;
        size_t tb_count = tk * tn;
        size_t tc_count = tm * tn;

        size_t ta_size = ta_count * sizeof(double);
        size_t tb_size = tb_count * sizeof(double);
        size_t tc_size = tc_count * sizeof(double);

        size_t h_a_size = ta_size * 3;
        size_t h_b_size = tb_size * 3;
        size_t h_c_size = tc_size * 3;

        double h_a_size_in_mb = (double) h_a_size / (1024*1024);
        double h_b_size_in_mb = (double) h_b_size / (1024*1024);
        double h_c_size_in_mb = (double) h_c_size / (1024*1024);
        double h_size_in_mb = h_a_size_in_mb + h_b_size_in_mb + h_c_size_in_mb;

     // Host Memory = alloc
        double * h_a = (double *) malloc( h_a_size );
        double * h_b = (double *) malloc( h_b_size );
        double * h_c = (double *) malloc( h_c_size );

     // Host Memory - register
        CUDA_RC_CHECK( hipHostRegister( h_a, h_a_size, 0 ) );
        CUDA_RC_CHECK( hipHostRegister( h_b, h_b_size, 0 ) );
        CUDA_RC_CHECK( hipHostRegister( h_c, h_c_size, 0 ) );


     // Device Memory - define
        size_t d_a_size = ta_count * sizeof(double) * 2;
        size_t d_b_size = tb_count * sizeof(double) * 2;
        size_t d_c_size = tc_count * sizeof(double) * 2;

        double d_a_size_in_mb = (double) d_a_size / (1024*1024);
        double d_b_size_in_mb = (double) d_b_size / (1024*1024);
        double d_c_size_in_mb = (double) d_c_size / (1024*1024);
        double d_size_in_mb = d_a_size_in_mb + d_b_size_in_mb + d_c_size_in_mb;

     // Device Memory - alloc
        double * d_a = NULL;
        double * d_b = NULL;
        double * d_c = NULL;
        CUDA_RC_CHECK( hipMalloc( (void **) &d_a, d_a_size ) );
        CUDA_RC_CHECK( hipMalloc( (void **) &d_b, d_b_size ) );
        CUDA_RC_CHECK( hipMalloc( (void **) &d_c, d_c_size ) );


     // Create CUDA Streams
        hipStream_t * stream = (hipStream_t *) malloc( sizeof(hipStream_t) * 2 );
        for(int i=0; i<2; i++) CUDA_RC_CHECK( hipStreamCreate( &stream[i] ) );

     // Prepare buffering pointers
        double * h_a_head = h_a;
        double * h_a_tail = h_a + ta_count*2;
        double * h_b_head = h_b;
        double * h_b_tail = h_b + tb_count*2;
        double * h_c_head = h_c;
        double * h_c_tail = h_c + tc_count*2;

        double * d_a_head = d_a;
        double * d_a_tail = d_a + ta_count;
        double * d_b_head = d_b;
        double * d_b_tail = d_b + tb_count;
        double * d_c_head = d_c;
        double * d_c_tail = d_c + tc_count;

        hipStream_t * stream_head = stream;
        hipStream_t * stream_tail = stream + 1;

        long ip = 0;            // number of patches streamed from the network
        size_t dlb_counter;       // dynamic load balancer

        if(me == 0) 
        {
           printf("\n");
           printf("---------------- Streaming DGEMM using DDI ---------------- \n"); 
           printf("global dimensions: %8d %8d %8d\n",m,n,k);
           printf("tile dimensions  : %8d %8d %8d\n",tm,tn,tk);
           printf("\n");
           printf("performing %d tiled dgemms over %d nodes\n", tiled_dgemm_count, nn);
           printf("host requirements   = %.2lf + %.2lf + %.2lf = %.2lf\n", h_a_size_in_mb,
                   h_b_size_in_mb, h_c_size_in_mb, h_size_in_mb);
           printf("device requirements = %.2lf + %.2lf + %.2lf = %.2lf\n", d_a_size_in_mb,
                   d_b_size_in_mb, d_c_size_in_mb, d_size_in_mb);
           fflush(stdout);
        }

        DDI_Sync(1234);

     // DDOT Algorithm - Start Up
     // Dynamically load-balance patches of C
     // Stream A & B from Distributed Memory 
     // C remains device resident

     // Start Up
     // Get load-balance counter
     // Convert counter to determine patch of C
        DDI_Patch a_patch, b_patch;
        size_t patch_count = m_patch_count * n_patch_count;
       
        double _alpha = 1.0;
        double _beta  = 1.0;
        double *alpha = &_alpha;
        double *beta  = &_beta;

        DDI_DLBNext(&dlb_counter);

        start_time = MPI_Wtime();
        while(dlb_counter < patch_count)
        {
           int patch_coord_i = dlb_counter / m_patch_count;
           int patch_coord_j = dlb_counter % m_patch_count;
   
        // Constant over these patch coordinates (i,j)
           a_patch.ilo = tm * patch_coord_i;
           a_patch.ihi = a_patch.ilo + tm - 1;
           b_patch.jlo = tn * patch_coord_j;
           b_patch.jhi = b_patch.jlo + tn - 1;
   
           for(int ik=0; ik<k_patch_count; ik++,ip++)
           {
               a_patch.jlo = tk * ik;
               a_patch.jhi = a_patch.jlo + tk - 1;
               b_patch.ilo = a_patch.jlo;
               b_patch.ihi = a_patch.jhi;
   
               if(ip > 1)
               {
                   CUBLAS_RC_CHECK( hipblasSetStream( cublas_hnd, *stream ) );
                   CUBLAS_RC_CHECK( hipblasDgemm( cublas_hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                                 tn, tm, tk, alpha, d_a, tn, d_b, tk, beta, d_c, tn ) );
                   if(stream == stream_tail) stream = stream_head;
                   else                      stream++;
                   if(d_a == d_a_tail) d_a = d_a_head;
                   else                d_a += ta_count;
                   if(d_b == d_b_tail) d_b = d_b_head;
                   else                d_b += tb_count;
               }
   
               if(ip > 0)
               {
                   hipStreamSynchronize( *stream );
                   /*
                   if(++dgemms_completed == k_patch_count) {
                      hipMemcpyAsync( h_c, d_c, c_size, hipMemcpyDeviceToHost, stream );
                      if(h_c == h_c_tail) h_c = h_c_head;
                      else                h_c += tc_count;
                      if(d_c == d_c_tail) d_c = d_c_head;
                      else                d_c += tc_count;
                      // zero out new d_c
                      dgemms_completed = 0;
                   }
                   */
                   CUDA_RC_CHECK( hipMemcpyAsync( d_a, h_a, ta_size, hipMemcpyHostToDevice, *stream ) );
                   CUDA_RC_CHECK( hipMemcpyAsync( d_b, h_b, tb_size, hipMemcpyHostToDevice, *stream ) );
                   if(h_a == h_a_tail) h_a = h_a_head;
                   else                h_a += ta_count;
                   if(h_b == h_b_tail) h_b = h_b_head;
                   else                h_b += tb_count;
               }

               DDI_GetP(ddi_a, &a_patch, h_a);
               DDI_GetP(ddi_b, &b_patch, h_b);

               // printf("end of loop %d\n", ip);
   
           } // end loop on ik

           DDI_DLBNext(&dlb_counter);

        } // end while loop


        // finished loops ==> drain buffers - step 1 of 2
        // moves last get on to the device & executes the dgemm for the data already there
        if(ip > 1)
        {
            CUBLAS_RC_CHECK( hipblasSetStream( cublas_hnd, *stream ) );
            CUBLAS_RC_CHECK( hipblasDgemm( cublas_hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                          tn, tm, tk, alpha, d_a, tn, d_b, tk, beta, d_c, tn ) );
            if(stream == stream_tail) stream = stream_head;
            else                      stream++;
            if(d_a == d_a_tail) d_a = d_a_head;
            else                d_a += ta_count;
            if(d_b == d_b_tail) d_b = d_b_head;
            else                d_b += tb_count;
        }
  
        if(ip > 0)
        {
            hipStreamSynchronize( *stream );
            /*
            if(++dgemms_completed == k_patch_count) {
               hipMemcpyAsync( h_c, d_c, c_size, hipMemcpyDeviceToHost, stream );
               if(h_c == h_c_tail) h_c = h_c_head;
               else                h_c += tc_count;
               if(d_c == d_c_tail) d_c = d_c_head;
               else                d_c += tc_count;
               // zero out new d_c
               dgemms_completed = 0;
            }
            */
            CUDA_RC_CHECK( hipMemcpyAsync( d_a, h_a, ta_size, hipMemcpyHostToDevice, *stream ) );
            CUDA_RC_CHECK( hipMemcpyAsync( d_b, h_b, tb_size, hipMemcpyHostToDevice, *stream ) );
            if(h_a == h_a_tail) h_a = h_a_head;
            else                h_a += ta_count;
            if(h_b == h_b_tail) h_b = h_b_head;
            else                h_b += tb_count;
        }

        // finished loops ==> drain buffers - step 2 of 2
        ++ip;

        if(ip > 1)
        {
            CUBLAS_RC_CHECK( hipblasSetStream( cublas_hnd, *stream ) );
            CUBLAS_RC_CHECK( hipblasDgemm( cublas_hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                          tn, tm, tk, alpha, d_a, tn, d_b, tk, beta, d_c, tn ) );
        }

        CUDA_RC_CHECK( hipMemcpyAsync( h_c, d_c, tc_size, hipMemcpyDeviceToHost, *stream ) );
        hipStreamSynchronize( *stream );

        DDI_Sync(1234);
        stop_time = MPI_Wtime();

        if(me == 0) {
           printf("walltime = %.6lf\n", (stop_time - start_time));
           fflush(stdout);
        }

     // Clean up memory
        DDI_Destroy(ddi_c);
        DDI_Destroy(ddi_b);
        DDI_Destroy(ddi_a);
        DDI_Finalize();
        return 0;
}
}

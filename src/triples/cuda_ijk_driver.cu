#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cuda_kernels.h"

//#define NO_CUDA_DEBUG
#ifndef NO_CUDA_DEBUG
#define CUDA_ERROR_CHECK()                                                                     \
{                                                                                              \
    hipError_t err = hipGetLastError();                                                      \
     if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
     printf(" +----------------------------------------\n"); \
     printf(" | ** CUDA ERROR! ** \n"); \
     printf(" | Error: \n"); \
     printf(" | Msg: %s\n", hipGetErrorString(err) ); \
     printf(" | File: %s\n", __FILE__ );  \
     printf(" | Line: %d\n", __LINE__ );  \
     printf(" +----------------------------------------\n"); \
                 exit(-1);  } \
}
#else
#define CUDA_ERROR_CHECK() {}
#endif

#define VM_INDEX(i,j)  ( (no*nu*no)*(j-1) + (no*nu)*(i-1) )
#define T2_INDEX(i)    ( (nu*nu*no)*(i-1) )
#define VOE_INDEX(i,j) ( (nu*nu*no)*(j-1) + (nu*nu)*(i-1) )

extern "C" {

#include "ddi.h"

typedef long Integer;

static long iold = -1;
static long jold = -1;
static long kold = -1;

static double *d_eh = NULL;
static double *d_ep = NULL;
static double *d_vm = NULL;
static double *d_v3 = NULL;
static double *d_ve_i = NULL;
static double *d_ve_j = NULL;
static double *d_ve_k = NULL;
static double *d_temp = NULL;
static double *d_t2   = NULL;
static double *d_voe  = NULL;
static double *d_t1   = NULL;
static double *d_x3   = NULL;
static double *d_etd_reduce = NULL;

static hipStream_t d_stream1, d_stream2;
static hipEvent_t  d_event_vej_exp;
static hipEvent_t  d_event_v3_free;
static hipblasHandle_t d_cublas;

static double *t1  = NULL;
static double *t2  = NULL;
static double *voe = NULL;

static double *ve_i = NULL;
static double *ve_j = NULL;
static double *ve_k = NULL;

void triples_cuda_init_(
        Integer *f_no,
        Integer *f_nu,
        double *f_eh,
        double *f_ep,
        double *f_t1,
        double *f_t2,
        double *f_vm,
        double *f_voe,
        double *f_ve_i,
        double *f_ve_j,
        double *f_ve_k)
{

        long no = (long) *f_no;
        long nu = (long) *f_nu;
        long no2 = no*no;
        long nu2 = nu*nu;
        long no3 = no*no*no;
        long nu3 = nu2*nu;
        long nutr = (nu2 + nu) / 2;
        long nou2 = no*nu*nu;
        long no2u2 = no2*nu2;
        size_t numbytes, gpu_bytes=0;

        hipError_t cudaStat;
        hipblasStatus_t stat;

     // d_eh
        numbytes = sizeof(double) * no; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_eh, numbytes );
        CUDA_ERROR_CHECK();
        hipMemcpy( d_eh, f_eh, numbytes, hipMemcpyHostToDevice );
        CUDA_ERROR_CHECK();

     // d_ep
        numbytes = sizeof(double) * nu; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_ep, numbytes );
        CUDA_ERROR_CHECK();
        hipMemcpy( d_ep, f_ep, numbytes, hipMemcpyHostToDevice );
        CUDA_ERROR_CHECK();

     // d_vm
        numbytes = sizeof(double) * no3 * nu; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_vm, numbytes );
        CUDA_ERROR_CHECK();
        hipMemcpy( d_vm, f_vm, numbytes, hipMemcpyHostToDevice );
        CUDA_ERROR_CHECK();

     // d_v3
        numbytes = sizeof(double) * nu3; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_v3, numbytes );
        CUDA_ERROR_CHECK();

     // d_ve_i
        numbytes = sizeof(double) * nu3; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_ve_i, numbytes );
        CUDA_ERROR_CHECK();

     // d_ve_j
        numbytes = sizeof(double) * nu3; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_ve_j, numbytes );
        CUDA_ERROR_CHECK();

     // d_ve_k
        numbytes = sizeof(double) * nu3; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_ve_k, numbytes );
        CUDA_ERROR_CHECK();

     // d_temp
        numbytes = sizeof(double) * nutr * nu;
        cudaStat = hipMalloc( (void **) &d_temp, numbytes );
        CUDA_ERROR_CHECK();

     // d_t2
        numbytes = sizeof(double) * nou2 * 3; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_t2, numbytes );
        CUDA_ERROR_CHECK();

     // d_voe
        numbytes = sizeof(double) * nu2 * 6; gpu_bytes += numbytes;
        cudaStat = hipMalloc( (void **) &d_voe, numbytes );
        CUDA_ERROR_CHECK();

     // d_t1
        t1 = f_t1;
        numbytes = sizeof(double) * nu * no;
        cudaStat = hipMalloc( (void **) &d_t1, numbytes );
        CUDA_ERROR_CHECK();
        hipMemcpy( d_t1, t1, numbytes, hipMemcpyHostToDevice );
        CUDA_ERROR_CHECK();

     // d_x3
        numbytes = sizeof(double);
        cudaStat = hipMalloc( (void **) &d_x3, numbytes );
        CUDA_ERROR_CHECK();
        hipMemset( d_x3, 0, numbytes );
        CUDA_ERROR_CHECK();

     // d_etd_reduce
        numbytes = sizeof(double) * nu * nu;
        cudaStat = hipMalloc( (void **) &d_etd_reduce, numbytes );
        CUDA_ERROR_CHECK();

     // cuda streams
        hipStreamCreate ( &d_stream1 );
        CUDA_ERROR_CHECK();
        hipStreamCreate ( &d_stream2 );
        CUDA_ERROR_CHECK();
       
     // cuda events
        hipEventCreate( &d_event_vej_exp );
        CUDA_ERROR_CHECK();
        hipEventCreate( &d_event_v3_free );
        CUDA_ERROR_CHECK();

     // cublas
        stat = hipblasCreate( &d_cublas );

     // host pointer to shared memory arrays - try to register them

     // t2
        t2  = f_t2;
        cudaStat = hipHostRegister( t2, no2u2, 0);
        if(cudaStat != hipSuccess) printf("hipHostRegister failed on t2 array.\n");
        CUDA_ERROR_CHECK();
        
     // voe
        voe = f_voe;
        cudaStat = hipHostRegister( voe, no2u2, 0);
        if(cudaStat != hipSuccess) printf("hipHostRegister failed for voe array.\n");
        CUDA_ERROR_CHECK();

     // ve_i, ve_j, ve_k
        ve_i = f_ve_i;
        ve_j = f_ve_j;
        ve_k = f_ve_k;
     /*
        double * ve = NULL;
        numbytes = sizeof(double) * nutr * nu * 3;
        //cudaStat = hipHostMalloc( (void **)&ve, numbytes );
        //CUDA_ERROR_CHECK();
        ve = (double *) malloc( numbytes );
        ve_i = ve;
        ve_j = ve + (nutr * nu);
        ve_j = ve + (nutr * nu)*2;
     */
}

void triples_cuda_finalize_(
        Integer *f_no,
        Integer *f_nu,
        double *f_x3)
{
        long no = *f_no;
        long nu = *f_nu;
        size_t numbytes; 

        hipFree( d_eh );
        CUDA_ERROR_CHECK();
        hipFree( d_ep );
        CUDA_ERROR_CHECK();
        hipFree( d_vm );
        CUDA_ERROR_CHECK();
        hipFree( d_v3 );
        CUDA_ERROR_CHECK();
        hipFree( d_ve_i );
        CUDA_ERROR_CHECK();
        hipFree( d_ve_j );
        CUDA_ERROR_CHECK();
        hipFree( d_ve_k );
        CUDA_ERROR_CHECK();
        hipFree( d_temp );
        CUDA_ERROR_CHECK();
        hipFree( d_t2 );
        CUDA_ERROR_CHECK();
        hipFree( d_voe );
        CUDA_ERROR_CHECK();

        hipStreamDestroy( d_stream1 );
        CUDA_ERROR_CHECK();
        hipStreamDestroy( d_stream2 );
        CUDA_ERROR_CHECK();

        hipEventDestroy( d_event_vej_exp );
        CUDA_ERROR_CHECK();
        hipEventDestroy( d_event_v3_free );
        CUDA_ERROR_CHECK();

        hipHostUnregister( t2 );
        CUDA_ERROR_CHECK();
        hipHostUnregister( voe );
        CUDA_ERROR_CHECK();

        numbytes = sizeof(double) * nu * no;
        hipMemcpy( t1, d_t1, numbytes, hipMemcpyDeviceToHost );
        CUDA_ERROR_CHECK();
        hipFree( d_t1 );
        CUDA_ERROR_CHECK();

        numbytes = sizeof(double);
        hipMemcpy( f_x3, d_x3, numbytes, hipMemcpyDeviceToHost );
        hipFree( d_x3 );
        CUDA_ERROR_CHECK();

        hipFree( d_etd_reduce );
        CUDA_ERROR_CHECK();
}

static DDI_Patch * ve_patch(long i, long nu, DDI_Patch * patch)
{
        long nutr = (nu*nu+nu)/2;
        patch->ilo = 0;
        patch->ihi = nutr-1;
        patch->jlo = nu*(i-1);
        patch->jhi = patch->jlo + nu;
        return patch;
}


void ijk_cuda_driver_(
    long int *p_nu, 
    long int *p_no,
    long int *p_i,
    long int *p_j,
    long int *p_k,
    double *ve_i,
    double *ve_j,
    double *ve_k)
{

  hipblasStatus_t stat;
  hipError_t cudaStat; 

  const double om = -1.0, zero = 0.0, one = 1.0;

  long int i = *p_i;
  long int j = *p_j;
  long int k = *p_k;

  long int no = *p_no;
  long int nu = *p_nu;
  long int nu2 = nu * nu;
  long int nu3 = nu2 * nu;
  long int nou2 = no * nu2;
  long int nutr = (nu2 + nu) / 2;

  double *t2_i, *t2_j, *t2_k;
  double *voe_ij, *voe_ji, *voe_ik, *voe_ki, *voe_jk, *voe_kj;
  double *d_t2_i, *d_t2_j, *d_t2_k;
  double *d_vm_ij, *d_vm_ji, *d_vm_ik, *d_vm_ki, *d_vm_kj, *d_vm_jk;
  double *d_voe_ij, *d_voe_ji, *d_voe_ik, *d_voe_ki, *d_voe_jk, *d_voe_kj;

  size_t numbytes;
  DDI_Patch patch;

/**
 * Determine VM offsets
 */
  d_vm_ij = d_vm + VM_INDEX(i,j);
  d_vm_ji = d_vm + VM_INDEX(j,i);
  d_vm_ik = d_vm + VM_INDEX(i,k);
  d_vm_ki = d_vm + VM_INDEX(k,i);
  d_vm_jk = d_vm + VM_INDEX(j,k);
  d_vm_kj = d_vm + VM_INDEX(k,j);

/**
 * Determine T2 offsets for the CPU and GPU
 */
  t2_i = t2 + T2_INDEX(i);
  t2_j = t2 + T2_INDEX(j);
  t2_k = t2 + T2_INDEX(k);
  d_t2_i = d_t2;
  d_t2_j = d_t2_i + nou2; 
  d_t2_k = d_t2_j + nou2;

/**
 * Determe VOE offsets for the CPU and GPU
 */
  voe_ij = voe + VOE_INDEX(i,j);
  voe_ji = voe + VOE_INDEX(j,i);
  voe_ik = voe + VOE_INDEX(i,k);
  voe_ki = voe + VOE_INDEX(k,i);
  voe_jk = voe + VOE_INDEX(j,k);
  voe_kj = voe + VOE_INDEX(k,j);
  d_voe_ij = d_voe;
  d_voe_ji = d_voe_ij + nu2;
  d_voe_ik = d_voe_ji + nu2;
  d_voe_ki = d_voe_ik + nu2;
  d_voe_jk = d_voe_ki + nu2;
  d_voe_kj = d_voe_jk + nu2;

/**
 * Set up grid / block for kernels
 */
  int blockx = 512;

  long int numblocks = ( nu3 / blockx ) + 1;

  dim3 block(blockx,1,1);
  long int gridx = 1;
  long int gridy = 1;

  if( numblocks <= 65535 )
  {
    gridx = numblocks;
  } else
  if( numblocks > 65535 && numblocks < (long int) 65535 * (long int )65535 )
  {
    gridx =  (long int) ceil( sqrt( (double) numblocks ) );
    gridy = gridx;
  } else
  {
    printf("too large grid requested...exiting\n");
    exit( 911 );
  } /* end if */

  dim3 grid( gridx, gridy, 1 );


/**
 * Form V3
 */

  if(j != jold) {
     numbytes = sizeof(double) * no * nu2;
     hipMemcpyAsync( d_t2_j, t2_j, numbytes, hipMemcpyHostToDevice, d_stream1 );
  }

  stat = hipblasSetStream( d_cublas, d_stream1 );
  hipStreamWaitEvent( d_stream1, d_event_v3_free, 0 );
  CUDA_ERROR_CHECK();
  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_j, nu2,
	   d_vm_ki, no, &zero,
	   d_v3, nu2 );

  if(k != kold) {
     numbytes = sizeof(double) * no * nu2;
     hipMemcpyAsync( d_t2_k, t2_k, numbytes, hipMemcpyHostToDevice, d_stream2 );
     CUDA_ERROR_CHECK();
  }

  if(j != jold) {
   # if HAVE_VE_EXPANSION_KERNEL
     //DDI_GetP(d_vvvo, ve_patch(j,nu,&patch), ve_j);
     numbytes = sizeof(double) * nutr * nu;
     hipMemcpyAsync( d_temp, ve_j, numbytes, hipMemcpyHostToDevice, d_stream2 );
     CUDA_ERROR_CHECK();
     exp_trsq_kernel<<< grid, block, 0, d_stream2 >>>( nu, d_temp, d_ve_j );
     CUDA_ERROR_CHECK();
     hipEventRecord( d_event_vej_exp, d_stream2 );
     CUDA_ERROR_CHECK();
     trant3_1_kernel<<< grid, block, 0, d_stream2 >>>( nu, d_ve_j );
     CUDA_ERROR_CHECK();
   # else
   # error "HAVE_VE_EXPANSION_KERNEL must be enabled"
   # endif
  }

  hipStreamSynchronize( d_stream1 );
  CUDA_ERROR_CHECK();
  stat = hipblasSetStream( d_cublas, d_stream2 );
  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_j, nu,
	   &d_t2_k[nu2*(i-1)], nu, &one,
	   d_v3, nu2 );

  trant3_1_kernel<<< grid, block, 0, d_stream2 >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ji, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );
  
  if(k != kold) {
   # if HAVE_VE_EXPANSION_KERNEL
  // d_ve_j must be expanded before d_temp can be reused
     //DDI_GetP(d_vvvo, ve_patch(k,nu,&patch), ve_k);
     hipStreamWaitEvent( d_stream1, d_event_vej_exp, 0 ); 
     CUDA_ERROR_CHECK();
     numbytes = sizeof(double) * nutr * nu;
     hipMemcpyAsync( d_temp, ve_k, numbytes, hipMemcpyHostToDevice, d_stream1 );
     CUDA_ERROR_CHECK();

     exp_trsq_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_temp, d_ve_k );
     CUDA_ERROR_CHECK();
     trant3_1_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_ve_k );
     CUDA_ERROR_CHECK();
   # endif
  }
  
  hipStreamSynchronize( d_stream2 );
  CUDA_ERROR_CHECK();
  stat = hipblasSetStream( d_cublas, d_stream1 );
  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_j[nu2*(i-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );

  trant3_4_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  if(i != iold) {
     numbytes = sizeof(double) * no * nu2;
     hipMemcpyAsync( d_t2_i, t2_i, numbytes, hipMemcpyHostToDevice, d_stream2 );
     CUDA_ERROR_CHECK();
   }

  hipStreamSynchronize( d_stream1 );
  CUDA_ERROR_CHECK();
  stat = hipblasSetStream( d_cublas, d_stream2 );
  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_i, nu2,
	   d_vm_kj, no, &one,
	   d_v3, nu2 );

  if(i != iold) {
   # if HAVE_VE_EXPANSION_KERNEL
  // no need to wait on a vek expand event (event_vek_exp), because it was done in stream1
     //DDI_GetP(d_vvvo, ve_patch(i,nu,&patch), ve_i);
     numbytes = sizeof(double) * nutr * nu;
     hipMemcpyAsync( d_temp, ve_i, numbytes, hipMemcpyHostToDevice, d_stream1 );
     CUDA_ERROR_CHECK();
     exp_trsq_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_temp, d_ve_i );
     CUDA_ERROR_CHECK();
     trant3_1_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_ve_i );
     CUDA_ERROR_CHECK();
   # endif
  }

  hipStreamSynchronize( d_stream2 );
  CUDA_ERROR_CHECK();
  stat = hipblasSetStream( d_cublas, d_stream1 );
  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_i, nu,
	   &d_t2_k[nu2*(j-1)], nu, &one,
	   d_v3, nu2 );

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_i[nu2*(k-1)], nu,
	   d_ve_j, nu, &one,
	   d_v3, nu );

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ik, no,
	   d_t2_j, nu2, &one,
	   d_v3, nu );

  trant3_1_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_i, nu2,
	   d_vm_jk, no, &one,
	   d_v3, nu2 );

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_i, nu,
	   &d_t2_j[nu2*(k-1)], nu, &one,
	   d_v3, nu2 );

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_i[nu2*(j-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );

  stat = hipblasDgemm( d_cublas,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ij, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );

  trant3_1_kernel<<< grid, block, 0, d_stream1 >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

/* 
 * final copy back of v3
 */

//  numbytes = sizeof(double) * nu3;
//  hipMemcpy( v3, d_v3, numbytes, hipMemcpyDeviceToHost );
//  CUDA_ERROR_CHECK();

//  hipFree( d_v3 );
//  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  hipMemcpyAsync( d_voe_ij, voe_ij, numbytes, hipMemcpyHostToDevice, d_stream2 );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  hipMemcpyAsync( d_voe_ji, voe_ji, numbytes, hipMemcpyHostToDevice, d_stream2 );
  CUDA_ERROR_CHECK();
  
  numbytes = sizeof(double) * nu2;
  hipMemcpyAsync( d_voe_ik, voe_ik, numbytes, hipMemcpyHostToDevice, d_stream2 );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  hipMemcpyAsync( d_voe_ki, voe_ki, numbytes, hipMemcpyHostToDevice, d_stream2 );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  hipMemcpyAsync( d_voe_jk, voe_jk, numbytes, hipMemcpyHostToDevice, d_stream2 );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  hipMemcpyAsync( d_voe_kj, voe_kj, numbytes, hipMemcpyHostToDevice, d_stream2 );
  CUDA_ERROR_CHECK();

  int device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties( &deviceProp, device );

//  const int blockx = deviceProp.warpSize * 6;
  block.x = SHARED_REDUCTION_SIZE;
//    block.x = 128;

//  printf("warpSize is %d\n",blockx);
  block.y = 1;


// Note if one changes gridx/gridy to be anything other than nu
// one must change the size of d_etd_reduce in ijk_gpu_init/finalize
  gridx = 1;
  gridy = 1;

  if( nu <= 65535 )
  {
    gridx = nu;
    gridy = nu;
  } else
  {
    printf("too large grid requested...exiting\n");
    exit( 911 );
  } /* end if */

  grid.x = gridx;
  grid.y = gridy;
  grid.z = 1;

/*
 * set the temporary array to zero it will be used for the reduction
 */
  numbytes = sizeof(double) * gridx * gridy;
  hipMemsetAsync( d_etd_reduce, 0, numbytes, d_stream2 );
  CUDA_ERROR_CHECK();

// ensure stream1 is finished ==> v3 is completely formed
  hipStreamSynchronize( d_stream1 );
  CUDA_ERROR_CHECK();

  etd_cuda_kernel<<< grid, block, 0, d_stream2 >>>( i, j, k, no, nu, d_v3,
       d_voe_ij, d_voe_ji, d_voe_ik, d_voe_ki, d_voe_jk, d_voe_kj, 
       d_t1, d_eh, d_ep, d_etd_reduce );
  CUDA_ERROR_CHECK();

  reduce_etd_kernel<<< 1, 1, 0, d_stream2 >>>( gridx * gridy, d_etd_reduce, d_x3 );
  CUDA_ERROR_CHECK();

  grid.x = nu;
  grid.y = 1;

  t1a_cuda_kernel<<< grid, block, 0, d_stream2 >>>( i, j, k, no, nu, d_v3,
       d_voe_ij, d_voe_ji, d_voe_ik, d_voe_ki, d_voe_jk, d_voe_kj, 
       d_t1, d_eh, d_ep, d_etd_reduce );
  CUDA_ERROR_CHECK();

  hipEventRecord( d_event_v3_free, d_stream2 );
  CUDA_ERROR_CHECK();

/**
 * Set iold, jold and kold
 */
  iold = i;
  jold = j;
  kold = k;

  return;
   
} /* end void */


static void ijk_lookup(int no, int ijk, int *i, int *j, int *k)
{
        int icntr = 0;
        for(int ii=0; ii<no; ii++)
        for(int jj=0; jj<ii; jj++)
        for(int kk=0; kk<jj; kk++) {
           if(icntr++ == ijk) {
              *i = ii+1;
              *j = jj+1;
              *k = kk+1;
              return;
           }
        }
}

void triples_cuda_cdriver_(
    long int *p_no,
    long int *p_nu, 
    long int *ijk_sr,
    long int *ijk_nr,
    long int *p_vvvo)
{
        int no = (int) *p_no;
        int nu = (int) *p_nu;
        int sr = (int) *ijk_sr;
        int nr = (int) *ijk_nr;
        int d_vvvo = (int) *p_vvvo;
        int ijk, i, j, k;

     // IJK Tuples
        for(ijk=sr; ijk<(sr+nr); ijk++)
        {
           ijk_lookup( no, ijk, &i, &j, &k );
           //ijk_cuda_driver(no, nu, i, j, k, d_vvvo);
        }

     // Work Steal IIJ / IJJ Tuples from CPU
     // This is a TODO
}

} /* end extern C */

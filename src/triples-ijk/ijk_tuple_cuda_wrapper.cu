#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "ijk_tuple_cuda_kernels.h"

//#define NO_CUDA_DEBUG
#ifndef NO_CUDA_DEBUG
#define CUDA_ERROR_CHECK()                                                                     \
{                                                                                              \
    hipError_t err = hipGetLastError();                                                      \
     if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
     printf(" +----------------------------------------\n"); \
     printf(" | ** CUDA ERROR! ** \n"); \
     printf(" | Error: \n"); \
     printf(" | Msg: %s\n", hipGetErrorString(err) ); \
     printf(" | File: %d\n", __FILE__);  \
     printf(" | Line: %d\n", __LINE__ );  \
     printf(" +----------------------------------------\n"); \
                 exit(-1);  } \
}
#else
#define CUDA_ERROR_CHECK() {}
#endif


extern "C" {
void ijk_tuple_cuda_wrapper_(
    long int *p_nu, 
    long int *p_no,
    long int *p_i,
    long int *p_j,
    long int *p_k,
    double *t2_i,
    double *t2_j,
    double *t2_k,
    double *vm_ij,
    double *vm_ji,
    double *vm_ik,
    double *vm_ki,
    double *vm_jk,
    double *vm_kj,
    double *ve_i,
    double *ve_j,
    double *ve_k,
    double *v3 )
{

  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipError_t cudaStat; 

  const double om = -1.0, zero = 0.0, one = 1.0;

  long int i = (*p_i); //fortran pointer offset
  long int no = *p_no;
  long int nu = *p_nu;
  long int nu2 = nu * nu;
  long int nu3 = nu2 * nu;

  double *d_t2_j, *d_vm_ki, *d_v3;
  double *d_ve_j, *d_ve_k, *d_t2_k, *d_vm_ji;

  size_t numbytes;

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_j, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_j, t2_j, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ki, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ki, vm_ki, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_v3, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_v3, v3, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasCreate( &handle );

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_j, nu2,
	   d_vm_ki, no, &zero,
	   d_v3, nu2 );
  
  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_j, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_j, ve_j, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_k, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_k, t2_k, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_j, nu,
	   &d_t2_k[nu2*(i-1)], nu, &one,
	   d_v3, nu2 );

  const int blockx = 512;

  dim3 block(blockx,1,1);
  dim3 grid( 
       (nu3 / block.x) % 65534 + 1,
       (nu3 / block.x) / 65534 + 1, 1);

//  printf("nu3 %d\n", nu3);

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);
  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();


  numbytes = sizeof(double) * nu * no;
  cudaStat = hipMalloc( (void **) &d_vm_ji, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ji, vm_ji, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ji, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );
  

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_k, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_k, ve_k, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  
  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_j[nu2*(i-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );


  trant3_4_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();



  numbytes = sizeof(double) * nu3;
  hipMemcpy( v3, d_v3, numbytes, hipMemcpyDeviceToHost );
  CUDA_ERROR_CHECK();

  hipFree( d_vm_ji );
  hipFree( d_vm_ki );
  hipFree( d_t2_j );
  hipFree( d_ve_j );
  hipFree( d_ve_k );
  hipFree( d_t2_k );
  hipFree( d_v3 );
  return;
   
} /* end void */
} /* end extern C */

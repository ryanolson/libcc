#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "ijk_tuple_cuda_kernels.h"

//#define NO_CUDA_DEBUG
#ifndef NO_CUDA_DEBUG
#define CUDA_ERROR_CHECK()                                                                     \
{                                                                                              \
    hipError_t err = hipGetLastError();                                                      \
     if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
     printf(" +----------------------------------------\n"); \
     printf(" | ** CUDA ERROR! ** \n"); \
     printf(" | Error: \n"); \
     printf(" | Msg: %s\n", hipGetErrorString(err) ); \
     printf(" | File: %s\n", __FILE__ );  \
     printf(" | Line: %d\n", __LINE__ );  \
     printf(" +----------------------------------------\n"); \
                 exit(-1);  } \
}
#else
#define CUDA_ERROR_CHECK() {}
#endif


extern "C" {
void ijk_tuple_cuda_wrapper_(
    long int *p_nu, 
    long int *p_no,
    long int *p_i,
    long int *p_j,
    long int *p_k,
    double *t2_i,
    double *t2_j,
    double *t2_k,
    double *vm_ij,
    double *vm_ji,
    double *vm_ik,
    double *vm_ki,
    double *vm_jk,
    double *vm_kj,
    double *ve_i,
    double *ve_j,
    double *ve_k,
    double *v3 )
{

  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipError_t cudaStat; 

  const double om = -1.0, zero = 0.0, one = 1.0;

  long int i = (*p_i); //fortran pointer offset
  long int j = (*p_j); //fortran pointer offset
  long int k = (*p_k); //fortran pointer offset
  long int no = *p_no;
  long int nu = *p_nu;
  long int nu2 = nu * nu;
  long int nu3 = nu2 * nu;

  double *d_t2_i, *d_t2_j, *d_t2_k;
  double *d_ve_i, *d_ve_j, *d_ve_k;
  double *d_vm_ij, *d_vm_ji, *d_vm_ik, *d_vm_ki, *d_vm_kj, *d_vm_jk;
  double *d_v3;

  size_t numbytes;

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_j, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_j, t2_j, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ki, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ki, vm_ki, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_v3, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_v3, v3, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasCreate( &handle );

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_j, nu2,
	   d_vm_ki, no, &zero,
	   d_v3, nu2 );
  
  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_j, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_j, ve_j, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_k, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_k, t2_k, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_j, nu,
	   &d_t2_k[nu2*(i-1)], nu, &one,
	   d_v3, nu2 );

  const int blockx = 512;

  dim3 block(blockx,1,1);
  dim3 grid( 
       (nu3 / block.x) % 65534 + 1,
       (nu3 / block.x) / 65534 + 1, 1);

//  printf("nu3 %d\n", nu3);

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);
  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();


  numbytes = sizeof(double) * nu * no;
  cudaStat = hipMalloc( (void **) &d_vm_ji, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ji, vm_ji, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ji, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );
  

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_k, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_k, ve_k, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  
  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_j[nu2*(i-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );


  trant3_4_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_i, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_i, t2_i, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_kj, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_kj, vm_kj, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_i, nu2,
	   d_vm_kj, no, &one,
	   d_v3, nu2 );


  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_i, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_i, ve_i, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_i, nu,
	   &d_t2_k[nu2*(j-1)], nu, &one,
	   d_v3, nu2 );


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_i[nu2*(k-1)], nu,
	   d_ve_j, nu, &one,
	   d_v3, nu );

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ik, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ik, vm_ik, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ik, no,
	   d_t2_j, nu2, &one,
	   d_v3, nu );


  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  
  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_jk, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_jk, vm_jk, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_i, nu2,
	   d_vm_jk, no, &one,
	   d_v3, nu2 );


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_i, nu,
	   &d_t2_j[nu2*(k-1)], nu, &one,
	   d_v3, nu2 );


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_i[nu2*(j-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ij, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ij, vm_ij, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ij, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );


  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

/* 
 * final copy back of v3
 */

  numbytes = sizeof(double) * nu3;
  hipMemcpy( v3, d_v3, numbytes, hipMemcpyDeviceToHost );
  CUDA_ERROR_CHECK();

  hipFree( d_vm_ij );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_ji );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_ik );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_ki );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_jk );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_kj );
  CUDA_ERROR_CHECK();
  hipFree( d_t2_i );
  CUDA_ERROR_CHECK();
  hipFree( d_t2_j );
  CUDA_ERROR_CHECK();
  hipFree( d_t2_k );
  CUDA_ERROR_CHECK();
  hipFree( d_ve_i );
  CUDA_ERROR_CHECK();
  hipFree( d_ve_j );
  CUDA_ERROR_CHECK();
  hipFree( d_ve_k );
  CUDA_ERROR_CHECK();
  hipFree( d_v3 );
  CUDA_ERROR_CHECK();

  return;
   
} /* end void */
} /* end extern C */

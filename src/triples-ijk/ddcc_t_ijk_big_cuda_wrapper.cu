#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cuda_kernels.h"

//#define NO_CUDA_DEBUG
#ifndef NO_CUDA_DEBUG
#define CUDA_ERROR_CHECK()                                                                     \
{                                                                                              \
    hipError_t err = hipGetLastError();                                                      \
     if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
     printf(" +----------------------------------------\n"); \
     printf(" | ** CUDA ERROR! ** \n"); \
     printf(" | Error: \n"); \
     printf(" | Msg: %s\n", hipGetErrorString(err) ); \
     printf(" | File: %s\n", __FILE__ );  \
     printf(" | Line: %d\n", __LINE__ );  \
     printf(" +----------------------------------------\n"); \
                 exit(-1);  } \
}
#else
#define CUDA_ERROR_CHECK() {}
#endif


extern "C" {
void ddcc_t_ijk_big_cuda_wrapper_(
    long int *p_nu, 
    long int *p_no,
    long int *p_i,
    long int *p_j,
    long int *p_k,
    double *t2_i,
    double *t2_j,
    double *t2_k,
    double *vm_ij,
    double *vm_ji,
    double *vm_ik,
    double *vm_ki,
    double *vm_jk,
    double *vm_kj,
    double *ve_i,
    double *ve_j,
    double *ve_k,
    double *voe_ij,
    double *voe_ji,
    double *voe_ik,
    double *voe_ki,
    double *voe_jk,
    double *voe_kj,
    double *t1,
    double *eh,
    double *ep,
    double *etd)
{

  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipError_t cudaStat; 

  const double om = -1.0, zero = 0.0, one = 1.0;

  long int i = (*p_i); //fortran pointer offset
  long int j = (*p_j); //fortran pointer offset
  long int k = (*p_k); //fortran pointer offset
  long int no = *p_no;
  long int nu = *p_nu;
  long int nu2 = nu * nu;
  long int nu3 = nu2 * nu;

  double *d_t2_i, *d_t2_j, *d_t2_k;
  double *d_ve_i, *d_ve_j, *d_ve_k;
  double *d_vm_ij, *d_vm_ji, *d_vm_ik, *d_vm_ki, *d_vm_kj, *d_vm_jk;
  double *d_v3;

  size_t numbytes;

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_j, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_j, t2_j, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ki, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ki, vm_ki, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_v3, numbytes );
  CUDA_ERROR_CHECK();

// this copy is unnecessary because d_v3 is the product with a beta=0.0
// hipMemcpy( d_v3, v3, numbytes, hipMemcpyHostToDevice );
// CUDA_ERROR_CHECK();

  stat = hipblasCreate( &handle );

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_j, nu2,
	   d_vm_ki, no, &zero,
	   d_v3, nu2 );
  
  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_j, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_j, ve_j, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_k, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_k, t2_k, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_j, nu,
	   &d_t2_k[nu2*(i-1)], nu, &one,
	   d_v3, nu2 );

  int blockx = 512;

  long int numblocks = ( nu3 / blockx ) + 1;

  dim3 block(blockx,1,1);
  long int gridx = 1;
  long int gridy = 1;

  if( numblocks <= 65535 )
  {
    gridx = numblocks;
  } else
  if( numblocks > 65535 && numblocks < (long int) 65535 * (long int )65535 )
  {
    gridx =  (long int) ceil( sqrt( (double) numblocks ) );
    gridy = gridx;
  } else
  {
    printf("too large grid requested...exiting\n");
    exit( 911 );
  } /* end if */

  dim3 grid( gridx, gridy, 1 );

//  printf("nu3 %d\n", nu3);

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);
  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();


  numbytes = sizeof(double) * nu * no;
  cudaStat = hipMalloc( (void **) &d_vm_ji, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ji, vm_ji, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ji, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );
  

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_k, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_k, ve_k, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  
  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_j[nu2*(i-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );

#if 1

  trant3_4_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2 * no;
  cudaStat = hipMalloc( (void **) &d_t2_i, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t2_i, t2_i, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_kj, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_kj, vm_kj, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_i, nu2,
	   d_vm_kj, no, &one,
	   d_v3, nu2 );


  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_ve_i, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ve_i, ve_i, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_i, nu,
	   &d_t2_k[nu2*(j-1)], nu, &one,
	   d_v3, nu2 );


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_i[nu2*(k-1)], nu,
	   d_ve_j, nu, &one,
	   d_v3, nu );

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ik, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ik, vm_ik, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ik, no,
	   d_t2_j, nu2, &one,
	   d_v3, nu );


  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();

  
  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_jk, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_jk, vm_jk, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu2, nu, no, &om,
	   d_t2_i, nu2,
	   d_vm_jk, no, &one,
	   d_v3, nu2 );


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu2, nu, nu, &one,
	   d_ve_i, nu,
	   &d_t2_j[nu2*(k-1)], nu, &one,
	   d_v3, nu2 );


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_N, HIPBLAS_OP_N,
	   nu, nu2, nu, &one,
	   &d_t2_i[nu2*(j-1)], nu,
	   d_ve_k, nu, &one,
	   d_v3, nu );

  numbytes = sizeof(double) * no * nu;
  cudaStat = hipMalloc( (void **) &d_vm_ij, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_vm_ij, vm_ij, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();


  stat = hipblasDgemm( handle,
           HIPBLAS_OP_T, HIPBLAS_OP_T,
	   nu, nu2, no, &om,
	   d_vm_ij, no,
	   d_t2_k, nu2, &one,
	   d_v3, nu );


  trant3_1_kernel<<< grid, block >>>( nu, d_v3 );
  CUDA_ERROR_CHECK();
#endif
/* 
 * final copy back of v3
 */

//  numbytes = sizeof(double) * nu3;
//  hipMemcpy( v3, d_v3, numbytes, hipMemcpyDeviceToHost );
//  CUDA_ERROR_CHECK();

  hipFree( d_vm_ij );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_ji );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_ik );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_ki );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_jk );
  CUDA_ERROR_CHECK();
  hipFree( d_vm_kj );
  CUDA_ERROR_CHECK();
  hipFree( d_t2_i );
  CUDA_ERROR_CHECK();
  hipFree( d_t2_j );
  CUDA_ERROR_CHECK();
  hipFree( d_t2_k );
  CUDA_ERROR_CHECK();
  hipFree( d_ve_i );
  CUDA_ERROR_CHECK();
  hipFree( d_ve_j );
  CUDA_ERROR_CHECK();
  hipFree( d_ve_k );
  CUDA_ERROR_CHECK();
//  hipFree( d_v3 );
//  CUDA_ERROR_CHECK();

  double x3;

  double *d_t1;
  double *d_voe_ij, *d_voe_ji, *d_voe_ik, *d_voe_ki, *d_voe_kj, *d_voe_jk;
  double *d_eh, *d_ep;
  double *d_x3, *d_etd_reduce;


//  numbytes = sizeof(double) * nu3;
//  cudaStat = hipMalloc( (void **) &d_v3, numbytes );
//  CUDA_ERROR_CHECK();
//  hipMemcpy( d_v3, v3, numbytes, hipMemcpyHostToDevice );
//  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ij, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ij, voe_ij, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ji, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ji, voe_ji, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();
  
  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ik, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ik, voe_ik, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ki, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ki, voe_ki, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_jk, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_jk, voe_jk, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_kj, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_kj, voe_kj, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu * no;
  cudaStat = hipMalloc( (void **) &d_t1, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t1, t1, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no;
  cudaStat = hipMalloc( (void **) &d_eh, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_eh, eh, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu;
  cudaStat = hipMalloc( (void **) &d_ep, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ep, ep, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double);
  cudaStat = hipMalloc( (void **) &d_x3, numbytes );
  CUDA_ERROR_CHECK();
  hipMemset( d_x3, 0, numbytes );
  CUDA_ERROR_CHECK();

  int device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties( &deviceProp, device );

//  const int blockx = deviceProp.warpSize * 6;
  block.x = SHARED_REDUCTION_SIZE;
//    block.x = 128;

//  printf("warpSize is %d\n",blockx);
  block.y = 1;


  gridx = 1;
  gridy = 1;

  if( nu <= 65535 )
  {
    gridx = nu;
    gridy = nu;
  } else
  {
    printf("too large grid requested...exiting\n");
    exit( 911 );
  } /* end if */

  grid.x = gridx;
  grid.y = gridy;
  grid.z = 1;

  numbytes = sizeof(double) * (gridx * gridy) ;
  cudaStat = hipMalloc( (void **) &d_etd_reduce, numbytes );
  CUDA_ERROR_CHECK();

/*
 * set the temporary array to zero it will be used for the reduction
 */

  hipMemset( d_etd_reduce, 0, numbytes );
  CUDA_ERROR_CHECK();

//  printf("nu %d\n", nu);

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);

  etd_cuda_kernel<<< grid, block >>>( i, j, k, no, nu, d_v3,
       d_voe_ij, d_voe_ji, d_voe_ik, d_voe_ki, d_voe_jk, d_voe_kj, 
       d_t1, d_eh, d_ep, d_etd_reduce );
  CUDA_ERROR_CHECK();

  reduce_etd_kernel<<<1,1>>>( gridx * gridy, d_etd_reduce, d_x3 );
  CUDA_ERROR_CHECK();

  grid.x = nu;
  grid.y = 1;

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);

  t1a_cuda_kernel<<< grid, block >>>( i, j, k, no, nu, d_v3,
       d_voe_ij, d_voe_ji, d_voe_ik, d_voe_ki, d_voe_jk, d_voe_kj, 
       d_t1, d_eh, d_ep, d_etd_reduce );
  CUDA_ERROR_CHECK();

/* 
 * final copy back of v3 and t1
 */

#if 1
  numbytes = sizeof(double) * nu * no;
  hipMemcpy( t1, d_t1, numbytes, hipMemcpyDeviceToHost );
  CUDA_ERROR_CHECK();
#endif
  numbytes = sizeof(double);
  hipMemcpy( &x3, d_x3, numbytes, hipMemcpyDeviceToHost );
  CUDA_ERROR_CHECK();

/*
 * no need to copy v3 back to host
 */
// numbytes = sizeof(double) * nu3;
// hipMemcpy( v3, d_v3, numbytes, hipMemcpyDeviceToHost );
// CUDA_ERROR_CHECK();

//  printf("C etd %e x3 %e\n",*etd,x3);

  if( i == j || j == k ) 
  {
    *etd = (*etd) + x3 * 0.5;
  } /* end if */
  else
  {
    *etd = (*etd) + x3;
  } /* end else */

  hipFree( d_voe_ij );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_ji );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_ik );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_ki );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_jk );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_kj );
  CUDA_ERROR_CHECK();
  hipFree( d_t1 );
  CUDA_ERROR_CHECK();
  hipFree( d_eh );
  CUDA_ERROR_CHECK();
  hipFree( d_ep );
  CUDA_ERROR_CHECK();
  hipFree( d_v3 );
  CUDA_ERROR_CHECK();
  hipFree( d_x3 );
  CUDA_ERROR_CHECK();
  hipFree( d_etd_reduce );
  CUDA_ERROR_CHECK();

  return;
   
} /* end void */
} /* end extern C */

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cuda_kernels.h"

#define NV_WARPSIZE 32

//#define NO_CUDA_DEBUG
#ifndef NO_CUDA_DEBUG
#define CUDA_ERROR_CHECK()                                                                     \
{                                                                                              \
    hipError_t err = hipGetLastError();                                                      \
     if ( err != hipSuccess && err != hipErrorSetOnActiveProcess ) { \
     printf(" +----------------------------------------\n"); \
     printf(" | ** CUDA ERROR! ** \n"); \
     printf(" | Error: \n"); \
     printf(" | Msg: %s\n", hipGetErrorString(err) ); \
     printf(" | File: %s\n", __FILE__ );  \
     printf(" | Line: %d\n", __LINE__ );  \
     printf(" +----------------------------------------\n"); \
                 exit(-1);  } \
}
#else
#define CUDA_ERROR_CHECK() {}
#endif


extern "C" {
void t1wt3_ijk_cuda_wrapper_(
    long int *p_i,
    long int *p_j,
    long int *p_k,
    long int *p_no,
    long int *p_nu, 
    double *v3,
    double *voe_ij,
    double *voe_ji,
    double *voe_ik,
    double *voe_ki,
    double *voe_jk,
    double *voe_kj,
    double *t1,
    double *eh,
    double *ep,
    double *etd)
{

  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipError_t cudaStat; 

  const double om = -1.0, zero = 0.0, one = 1.0;

  long int i = (*p_i); //fortran pointer offset
  long int j = (*p_j); //fortran pointer offset
  long int k = (*p_k); //fortran pointer offset
  long int no = *p_no;
  long int nu = *p_nu;
  long int nu2 = nu * nu;
  long int nu3 = nu2 * nu;

  double x3;

  double *d_t1;
  double *d_voe_ij, *d_voe_ji, *d_voe_ik, *d_voe_ki, *d_voe_kj, *d_voe_jk;
  double *d_v3;
  double *d_eh, *d_ep;
  double *d_x3, *d_etd_reduce;

  size_t numbytes;

  numbytes = sizeof(double) * nu3;
  cudaStat = hipMalloc( (void **) &d_v3, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_v3, v3, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ij, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ij, voe_ij, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ji, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ji, voe_ji, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();
  
  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ik, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ik, voe_ik, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_ki, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_ki, voe_ki, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_jk, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_jk, voe_jk, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu2;
  cudaStat = hipMalloc( (void **) &d_voe_kj, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_voe_kj, voe_kj, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu * no;
  cudaStat = hipMalloc( (void **) &d_t1, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_t1, t1, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * no;
  cudaStat = hipMalloc( (void **) &d_eh, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_eh, eh, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double) * nu;
  cudaStat = hipMalloc( (void **) &d_ep, numbytes );
  CUDA_ERROR_CHECK();
  hipMemcpy( d_ep, ep, numbytes, hipMemcpyHostToDevice );
  CUDA_ERROR_CHECK();

  numbytes = sizeof(double);
  cudaStat = hipMalloc( (void **) &d_x3, numbytes );
  CUDA_ERROR_CHECK();
  hipMemset( d_x3, 0, numbytes );
  CUDA_ERROR_CHECK();

  int device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties( &deviceProp, device );

//  const int blockx = deviceProp.warpSize * 6;
  const int blockx = SHARED_REDUCTION_SIZE;
//  const int blockx = 32;

//  printf("warpSize is %d\n",blockx);
  const int blocky = 1;

  dim3 block(blockx,blocky,1);

  long int gridx = 1;
  long int gridy = 1;

  if( nu <= 65535 )
  {
    gridx = nu;
    gridy = nu;
  } else
  {
    printf("too large grid requested...exiting\n");
    exit( 911 );
  } /* end if */

  dim3 grid( gridx, gridy, 1 );

  numbytes = sizeof(double) * (gridx * gridy) ;
  cudaStat = hipMalloc( (void **) &d_etd_reduce, numbytes );
  CUDA_ERROR_CHECK();

/*
 * set the temporary array to zero it will be used for the reduction
 */

  hipMemset( d_etd_reduce, 0, numbytes );
  CUDA_ERROR_CHECK();

//  printf("nu %d\n", nu);

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);

  etd_cuda_kernel<<< grid, block >>>( i, j, k, no, nu, d_v3,
       d_voe_ij, d_voe_ji, d_voe_ik, d_voe_ki, d_voe_jk, d_voe_kj, 
       d_t1, d_eh, d_ep, d_etd_reduce );
  CUDA_ERROR_CHECK();

  reduce_etd_kernel<<<1,1>>>( gridx * gridy, d_etd_reduce, d_x3 );
  CUDA_ERROR_CHECK();

  grid.x = nu;
  grid.y = 1;

//  printf("block x y z %d %d %d\n",block.x,block.y,block.z);
//  printf("grid x y z %d %d %d\n",grid.x,grid.y,grid.z);

  t1a_cuda_kernel<<< grid, block >>>( i, j, k, no, nu, d_v3,
       d_voe_ij, d_voe_ji, d_voe_ik, d_voe_ki, d_voe_jk, d_voe_kj, 
       d_t1, d_eh, d_ep, d_etd_reduce );
  CUDA_ERROR_CHECK();

/* 
 * final copy back of v3 and t1
 */

#if 1
  numbytes = sizeof(double) * nu * no;
  hipMemcpy( t1, d_t1, numbytes, hipMemcpyDeviceToHost );
  CUDA_ERROR_CHECK();
#endif
  numbytes = sizeof(double);
  hipMemcpy( &x3, d_x3, numbytes, hipMemcpyDeviceToHost );
  CUDA_ERROR_CHECK();

//  printf("C etd %e x3 %e\n",*etd,x3);

  if( i == j || j == k ) 
  {
    *etd = (*etd) + x3 * 0.5;
  } /* end if */
  else
  {
    *etd = (*etd) + x3;
  } /* end else */

  hipFree( d_voe_ij );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_ji );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_ik );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_ki );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_jk );
  CUDA_ERROR_CHECK();
  hipFree( d_voe_kj );
  CUDA_ERROR_CHECK();
  hipFree( d_t1 );
  CUDA_ERROR_CHECK();
  hipFree( d_eh );
  CUDA_ERROR_CHECK();
  hipFree( d_ep );
  CUDA_ERROR_CHECK();
  hipFree( d_v3 );
  CUDA_ERROR_CHECK();
  hipFree( d_x3 );
  CUDA_ERROR_CHECK();
  hipFree( d_etd_reduce );
  CUDA_ERROR_CHECK();

  return;
   
} /* end void */
} /* end extern C */

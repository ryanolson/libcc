#include "hip/hip_runtime.h"
#include "stdio.h"

#define INDX(a,b,c,ld) ( ( (c) * (ld) * (ld) ) \
                       + ( (b) * (ld) ) \
                       + ( (a) ) )

extern "C" {

__global__ void exp_trsq_kernel( long int n, double *src, double *dest )
{

  int n32, a, b, c, myindx;

  n32 = (int) n;
  int nu = n32;

  int nu2 = nu * nu;
  int nutr = (nu2 + nu) / 2;

  myindx = ( blockIdx.z * ( gridDim.x * gridDim.y ) 
           + blockIdx.y * ( gridDim.x ) 
	   + blockIdx.x ) 
           * blockDim.x
	   + threadIdx.x;

  if( myindx >= ( n32 * n32 * n32 )  ) return;
  
  a = ( myindx % ( n32 * n32 ) ) % n32;
  b = ( myindx % ( n32 * n32 ) ) / n32;
  c =   myindx / ( n32 * n32 );

  int row = min( a, b );
  int col = max( a, b );

  int offset = c * nutr;

  offset += ( col * (col + 1) / 2);
  offset += row;

    dest[INDX(a, b, c, n32)] = src[offset];

} /* end exp_trsq_kernel */

__global__ void expand_tr_kernel( long int n, double *v )
{
  int nu = (int) n;
  int nu2 = nu * nu;
  int ij = ( ( ( nu2 + nu ) / 2 ) * nu ) - 1;

  for( int kloop = nu - 1; kloop >= 0; kloop-- )
  {
    int koff = kloop * nu2;
    for( int iloop = nu - 1; iloop >= 0; iloop-- )
    {
      for( int jloop = iloop; jloop >= 0; jloop-- )
      {
        int joff = iloop*nu + jloop;
//        v[koff + joff] = v[ij];
        v[INDX(jloop, iloop, kloop, nu)] = v[ij];
        ij--;
      } /* end for */
    } /* end for */
  } /* end for */

  return; 
} /* end expand_tr */

__global__ void expand_trsq_kernel( long int n, double *v )
{
  int n32, a, b, c, myindx;

  n32 = (int) n;

  myindx = ( blockIdx.z * ( gridDim.x * gridDim.y ) 
           + blockIdx.y * ( gridDim.x ) 
	   + blockIdx.x ) 
           * blockDim.x
	   + threadIdx.x;

  if( myindx >= ( n32 * n32 * n32 )  ) return;
  
  a = ( myindx % ( n32 * n32 ) ) % n32;
  b = ( myindx % ( n32 * n32 ) ) / n32;
  c =   myindx / ( n32 * n32 );

  if( b > a ) return;

  v[INDX(a, b, c, n32)] = v[INDX(b, a, c, n32)];

  return;

} /* end expand_trsq */

__global__ void trant3_1_kernel( long int n, double *v )
{
  int ltr, n32, a, b, c, myindx;
  double temp;

  n32 = (int) n;
  ltr = ( n*n - n ) / 2;

  myindx = ( blockIdx.z * ( gridDim.x * gridDim.y ) 
           + blockIdx.y * ( gridDim.x ) 
	   + blockIdx.x ) 
           * blockDim.x
	   + threadIdx.x;

  if( myindx >= ( n32 * n32 * n32 )  ) return;
  
  a = ( myindx % ( n32 * n32 ) ) % n32;
  b = ( myindx % ( n32 * n32 ) ) / n32;
  c =   myindx / ( n32 * n32 );

//  printf("tidx %d bidx %d bidy %d myindex %d a %d b %d c %d\n",
 //     threadIdx.x, blockIdx.x, blockIdx.y, myindx, a, b, c );

  if ( c >= b ) return;

  temp = v[INDX(a, b, c, n32)];
  v[INDX(a, b, c, n32)] = v[INDX(a, c, b, n32)];
  v[INDX(a, c, b, n32)] = temp;

  return;

} /* end trant3_1_kernel */

__global__ void trant3_4_kernel( long int n, double *v )
{
  int n32, a, b, c, myindx;
  double temp;

  n32 = (int) n;

  myindx = ( blockIdx.z * ( gridDim.x * gridDim.y )
           + blockIdx.y * ( gridDim.x )
           + blockIdx.x )
           * blockDim.x
           + threadIdx.x;

#if 1
  if( myindx >= ( n32 * n32 * n32 )  ) return;

  a = ( myindx % ( n32 * n32 ) ) % n32;
  b = ( myindx % ( n32 * n32 ) ) / n32;
  c =   myindx / ( n32 * n32 );

//  printf("tidx %d bidx %d bidy %d myindex %d a %d b %d c %d\n",
 //     threadIdx.x, blockIdx.x, blockIdx.y, myindx, a, b, c );

  if( c > b ) return;
  if( a > c ) return;

  temp = v[INDX(a, b, c, n32)];
  v[INDX(a, b, c, n32)] = v[INDX(b, c, a, n32)];
  v[INDX(b, c, a, n32)] = v[INDX(c, a, b, n32)];
  v[INDX(c, a, b, n32)] = temp;

  if( ( b == c ) || ( c == a ) ) return;

  temp = v[INDX(b, a, c, n32)];
  v[INDX(b, a, c, n32)] = v[INDX(a, c, b, n32)];
  v[INDX(a, c, b, n32)] = v[INDX(c, b, a, n32)];
  v[INDX(c, b, a, n32)] = temp;

  return;
#endif
#if 0
  if( myindx > 0 ) return;
  printf("tidx %d bidx %d bidy %d myindex %d a %d b %d c %d\n",
      threadIdx.x, blockIdx.x, blockIdx.y, myindx, a, b, c );
  for( b = 0; b < n32; b++ )
  {
    for( c = 0; c <= b; c++ )
    {
      for( a = 0; a <= c; a++ )
      {
        temp = v[INDX(a, b, c, n32)];
        v[INDX(a, b, c, n32)] = v[INDX(b, c, a, n32)];
        v[INDX(b, c, a, n32)] = v[INDX(c, a, b, n32)];
        v[INDX(c, a, b, n32)] = temp;
        if( b == c || c == a ) {}
	else
        {
          temp = v[INDX(b, a, c, n32)];
          v[INDX(b, a, c, n32)] = v[INDX(a, c, b, n32)];
          v[INDX(a, c, b, n32)] = v[INDX(c, b, a, n32)];
          v[INDX(c, b, a, n32)] = temp;
        }
      }
    }
  }
  return;
#endif
} /* end trant3_4_kernel */

} /* end extern C */

#include "hip/hip_runtime.h"
//#define INDX(a,b,c,ld) ( ( (c) * (ld) * (ld) ) \
                       + ( (b) * (ld) ) \
                       + ( (a) ) )

//extern "C" {

__device__ void warpReduce( volatile double *sdata )
{
#if 0
  if( blockDim.x >= 64 && blockDim.x % 32 == 0 )
  {
    for( unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
      if( threadIdx.x < s ) 
         sdata[threadIdx.x] += sdata[threadIdx.x + s];
      __syncthreads();
    } /* end for */
    if( threadIdx.x < 32 ) 
    {
      sdata[threadIdx.x] += sdata[threadIdx.x + 32];
      sdata[threadIdx.x] += sdata[threadIdx.x + 16];
      sdata[threadIdx.x] += sdata[threadIdx.x +  8];
      sdata[threadIdx.x] += sdata[threadIdx.x +  4];
      sdata[threadIdx.x] += sdata[threadIdx.x +  2];
      sdata[threadIdx.x] += sdata[threadIdx.x +  1];
    } /* end if */
  }
  else
#endif
  {
    if( threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 )
    {
      for( int idx = 1; idx < blockDim.x * blockDim.y * blockDim.z; idx++ )
      {
        sdata[0] += sdata[idx];
      } /* end for */
    } /* end if */
  } /* end */
} /* end warp Reduce */

__global__ void etd_cuda_kernel( const int i, const int j, const int k, 
        const int no, const int nu,
        const double *v3, const double *voe_ij, const double *voe_ji, 
	const double *voe_ik, const double *voe_ki, 
	const double *voe_jk, const double *voe_kj,
        double *t1, const double *eh, const double *ep, double *etd_reduce )
{

__shared__ double etd_shared[SHARED_REDUCTION_SIZE];

  int a = blockIdx.x;
  int b = blockIdx.y;
  double dijk = eh[i-1] + eh[j-1] + eh[k-1];
  double x3 = 0.0;
  const double two = 2.0, four = 4.0, eight = 8.0;
  double d1,d2,d3,f;

  for( int idx = 0; idx < nu; idx += blockDim.x )
  {
    int c = idx + threadIdx.x;

/*
 * don't do the loop if my id is outside the bounds of nu
 */

    if( c < nu )
    {

      if( a > b ) goto loop_end;
      if( a == b && b == c ) goto loop_end;
      double dabc = ep[a] + ep[b] + ep[c];
      double denom = 1.0 / ( dijk - dabc );

#ifdef NOTEX
      double abc = v3[ INDX(a, b, c, nu) ];
      double acb = v3[ INDX(a, c, b, nu) ];
      double bac = v3[ INDX(b, a, c, nu) ];
      double bca = v3[ INDX(b, c, a, nu) ];
      double cba = v3[ INDX(c, b, a, nu) ];
      double cab = v3[ INDX(c, a, b, nu) ];

      d1 = abc;
      d2 = acb + cba + bac;
      d3 = bca + cab;
      f  = d1*eight - d2*four + d3*two;
      x3        += f*d1*denom;
#if 0
      d1 = v3[INDX(a, b, c, nu)];
      d2 = v3[INDX(a, c, b, nu)] + v3[INDX(c, b, a, nu)] 
	        + v3[INDX(b, a, c, nu)];
      d3 = v3[INDX(b, c, a, nu)] + v3[INDX(c, a, b, nu)];
      f  = d1*eight - d2*four + d3*two;
      x3        += f*d1*denom;
#endif
#else
      double abc = fetch_x_v3( INDX(a, b, c, nu) );
      double acb = fetch_x_v3( INDX(a, c, b, nu) );
      double bac = fetch_x_v3( INDX(b, a, c, nu) );
      double bca = fetch_x_v3( INDX(b, c, a, nu) );
      double cba = fetch_x_v3( INDX(c, b, a, nu) );
      double cab = fetch_x_v3( INDX(c, a, b, nu) );

      d1 = abc;
      d2 = acb + cba + bac;
      d3 = bca + cab;
      f  = d1*eight - d2*four + d3*two;
      x3        += f*d1*denom;

//      d1 = fetch_x_v3( INDX(a, b, c, nu) );
//      d2 = fetch_x_v3( INDX(a, c, b, nu) ) + fetch_x_v3( INDX(c, b, a, nu) )
//	        + fetch_x_v3( INDX(b, a, c, nu) );
//      d3 = fetch_x_v3( INDX(b, c, a, nu) ) + fetch_x_v3( INDX(c, a, b, nu) );
//      f  = d1*eight - d2*four + d3*two;
//      x3        += f*d1*denom;
#endif

      if( a == b ) goto loop_end;

#ifdef NOTEX
      d1 = bac;
      d2 = bca + cab + abc;
      d3 = acb + cba;
      f  = d1*eight - d2*four + d3*two;
      x3 += f*d1*denom;
#if 0
      d1 = v3[INDX(b, a, c, nu)];
      d2 = v3[INDX(b, c, a, nu)] + v3[INDX(c, a, b, nu)] 
	 + v3[INDX(a, b, c, nu)];
      d3 = v3[INDX(a, c, b, nu)] + v3[INDX(c, b, a, nu)];
      f  = d1*eight - d2*four + d3*two;
      x3 += f*d1*denom;
#endif
#else
      d1 = bac;
      d2 = bca + cab + abc;
      d3 = acb + cba;
      f  = d1*eight - d2*four + d3*two;
      x3 += f*d1*denom;

//      d1 = fetch_x_v3( INDX(b, a, c, nu) );
//      d2 = fetch_x_v3( INDX(b, c, a, nu) ) + fetch_x_v3( INDX(c, a, b, nu) )
//	 + fetch_x_v3( INDX(a, b, c, nu) );
//      d3 = fetch_x_v3( INDX(a, c, b, nu) ) + fetch_x_v3( INDX(c, b, a, nu) );
//      f  = d1*eight - d2*four + d3*two;
//      x3 += f*d1*denom;
#endif

    } /* end if */
    
loop_end:
  } /* end for */


  etd_shared[threadIdx.x] = 0.0;
  etd_shared[threadIdx.x] = x3;

  __syncthreads();

  int offset = INDX(a, b, 0, gridDim.x );

#if 0
  if( threadIdx.x == 0 )
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      temp += etd_shared[idx];
    } /* end for */
    etd_reduce[offset] = temp;
  } /* end if */
#endif
#if 0
  for( unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
  {
    if( threadIdx.x < s ) 
       etd_shared[threadIdx.x] += etd_shared[threadIdx.x + s];
    __syncthreads();
  } /* end for */

  if( threadIdx.x < 32 ) warpReduce( etd_shared, threadIdx.x );

  if( threadIdx.x == 0 ) etd_reduce[offset] = etd_shared[0];
#endif
#if 1
  warpReduce( etd_shared );
#endif
  __syncthreads();
  if( threadIdx.x == 0 ) etd_reduce[offset] = etd_shared[0];



//  etd_shared[threadIdx.x] = t1ai;

} /* end kernel */


__global__ void t1a_cuda_kernel( const int i, const int j, const int k, 
        const int no, const int nu,
        const double *v3, const double *voe_ij, const double *voe_ji, 
	const double *voe_ik, const double *voe_ki, 
	const double *voe_jk, const double *voe_kj,
        double *t1, const double *eh, const double *ep, double *etd_reduce )
{

__shared__ double etd_shared[SHARED_REDUCTION_SIZE];

  int a = blockIdx.x;
  int b;
  double dijk = eh[i-1] + eh[j-1] + eh[k-1];
  const double two = 2.0, om = -1.0;
  double t1ai = 0.0, t1aj = 0.0, t1ak = 0.0;
  double t1bi = 0.0, t1bj = 0.0, t1bk = 0.0;
  int ti1d = threadIdx.z * ( blockDim.x * blockDim.y )
           + threadIdx.y * ( blockDim.x ) 
	   + threadIdx.x;

  for( b = 0; b < nu; b++ )
  { 
    for( int idx = 0; idx < nu; idx += ( blockDim.x * blockDim.y * blockDim.z) )
    {
      int c = idx + ti1d;

/*
 * don't do the loop if my id is outside the bounds of nu
 */

      if( c < nu )
      {
        if( a > b ) goto loop_end;
        if( a == b && b == c ) goto loop_end;
        double dabc = ep[a] + ep[b] + ep[c];
        double denom = 1.0 / ( dijk - dabc );
        if( a == b ) goto loop_end;

#ifdef NOTEX
        double abc = v3[ INDX(a, b, c, nu) ];
        double acb = v3[ INDX(a, c, b, nu) ];
        double bac = v3[ INDX(b, a, c, nu) ];
        double bca = v3[ INDX(b, c, a, nu) ];
        double cba = v3[ INDX(c, b, a, nu) ];
        double cab = v3[ INDX(c, a, b, nu) ];

        double t3_ab1 = ( abc - bac ) * two
   	              -   acb + bca;

        double t3_ab2 = ( acb - bca ) * two
	              -   abc + bac;

        double t3_ab3 = ( bac - abc ) * two
   	              -   cab + cba;

        double t3_ab5 = ( cab - cba )  * two
	              -   bac + abc;

        double t3_ab4 = ( bca - acb ) * two
	              -   cba + cab;

        double t3_ab6 = ( cba - cab ) * two
	              -   bca + acb;
#if 0
        double t3_ab1 = ( v3[INDX(a,b,c,nu)] - v3[INDX(b,a,c,nu)] ) * two
   	              -   v3[INDX(a,c,b,nu)] + v3[INDX(b,c,a,nu)];

        double t3_ab2 = ( v3[INDX(a,c,b,nu)] - v3[INDX(b,c,a,nu)] ) * two
	              -   v3[INDX(a,b,c,nu)] + v3[INDX(b,a,c,nu)];

        double t3_ab3 = ( v3[INDX(b,a,c,nu)] - v3[INDX(a,b,c,nu)] ) * two
   	              -   v3[INDX(c,a,b,nu)] + v3[INDX(c,b,a,nu)];

        double t3_ab5 = ( v3[INDX(c,a,b,nu)] - v3[INDX(c,b,a,nu)] ) * two
	              -   v3[INDX(b,a,c,nu)] + v3[INDX(a,b,c,nu)];

        double t3_ab4 = ( v3[INDX(b,c,a,nu)] - v3[INDX(a,c,b,nu)] ) * two
	              -   v3[INDX(c,b,a,nu)] + v3[INDX(c,a,b,nu)];

        double t3_ab6 = ( v3[INDX(c,b,a,nu)] - v3[INDX(c,a,b,nu)] ) * two
	              -   v3[INDX(b,c,a,nu)] + v3[INDX(a,c,b,nu)];
#endif
#else
        double abc = fetch_x_v3( INDX(a, b, c, nu) );
        double acb = fetch_x_v3( INDX(a, c, b, nu) );
        double bac = fetch_x_v3( INDX(b, a, c, nu) );
        double bca = fetch_x_v3( INDX(b, c, a, nu) );
        double cba = fetch_x_v3( INDX(c, b, a, nu) );
        double cab = fetch_x_v3( INDX(c, a, b, nu) );

        double t3_ab1 = ( abc - bac ) * two
   	              -   acb + bca;

        double t3_ab2 = ( acb - bca ) * two
	              -   abc + bac;

        double t3_ab3 = ( bac - abc ) * two
   	              -   cab + cba;

        double t3_ab5 = ( cab - cba )  * two
	              -   bac + abc;

        double t3_ab4 = ( bca - acb ) * two
	              -   cba + cab;

        double t3_ab6 = ( cba - cab ) * two
	              -   bca + acb;
#endif



//      if( a == 0 && b == 4 ) printf("c %d t3_ab1 %22.17e t3_ab2 %22.17e\n",
//	   c,t3_ab1,t3_ab2);

        t1ai += ( t3_ab1*voe_jk[INDX(b,c,0,nu)] 
	     +    t3_ab2*voe_kj[INDX(b,c,0,nu)] ) * denom;

        t1aj += ( t3_ab3*voe_ik[INDX(b,c,0,nu)] 
	     +    t3_ab5*voe_ki[INDX(b,c,0,nu)] ) * denom;

        t1ak += ( t3_ab4*voe_ij[INDX(b,c,0,nu)] 
	     +    t3_ab6*voe_ji[INDX(b,c,0,nu)] ) * denom;

      } /* end if */

loop_end:

    } /* end idx loop */

  } /* end b loop */


  etd_shared[ti1d] = t1ai;

  __syncthreads();

  int offi = INDX(a,i-1,0,nu);
#if 0
  if( ti1d == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempi += etd_shared[idx];
    } /* end for */
    t1[offi] += tempi;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( ti1d == 0 ) t1[offi] += etd_shared[0];

  __syncthreads();

  etd_shared[ti1d] = t1aj;

  __syncthreads();

  int offj = INDX(a,j-1,0,nu);
#if 0
  if( ti1d == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempj += etd_shared[idx];
    } /* end for */
    t1[offj] += tempj;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( ti1d == 0 ) t1[offj] += etd_shared[0];

  __syncthreads();

  etd_shared[ti1d] = t1ak;

  __syncthreads();

  int offk = INDX(a,k-1,0,nu);
#if 0
  if( ti1d == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempk += etd_shared[idx];
    } /* end for */
    t1[offk] += tempk;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( ti1d == 0 ) t1[offk] += etd_shared[0];

#if 1

  b = blockIdx.x;

  for( a = 0; a < nu; a++ )
  { 
    for( int idx = 0; idx < nu; idx += ( blockDim.x * blockDim.y * blockDim.z ) )
    {
      int c = idx + ti1d;

/*
 * don't do the loop if my id is outside the bounds of nu
 */

      if( c < nu )
      {
        if( a > b ) goto loop_end1;
        if( a == b && b == c ) goto loop_end1;
        double dabc = ep[a] + ep[b] + ep[c];
        double denom = 1.0 / ( dijk - dabc );
        if( a == b ) goto loop_end1;

#ifdef NOTEX
        double abc = v3[ INDX(a, b, c, nu) ];
        double acb = v3[ INDX(a, c, b, nu) ];
        double bac = v3[ INDX(b, a, c, nu) ];
        double bca = v3[ INDX(b, c, a, nu) ];
        double cba = v3[ INDX(c, b, a, nu) ];
        double cab = v3[ INDX(c, a, b, nu) ];

        double t3_ab1 = ( abc - bac ) * two
   	              -   acb + bca;

        double t3_ab2 = ( acb - bca ) * two
	              -   abc + bac;

        double t3_ab3 = ( bac - abc ) * two
   	              -   cab + cba;

        double t3_ab5 = ( cab - cba )  * two
	              -   bac + abc;

        double t3_ab4 = ( bca - acb ) * two
	              -   cba + cab;

        double t3_ab6 = ( cba - cab ) * two
	              -   bca + acb;
#if 0
        double t3_ab1 = ( v3[INDX(a,b,c,nu)] - v3[INDX(b,a,c,nu)] ) * two
   	              -   v3[INDX(a,c,b,nu)] + v3[INDX(b,c,a,nu)];

        double t3_ab2 = ( v3[INDX(a,c,b,nu)] - v3[INDX(b,c,a,nu)] ) * two
	              -   v3[INDX(a,b,c,nu)] + v3[INDX(b,a,c,nu)];

        double t3_ab3 = ( v3[INDX(b,a,c,nu)] - v3[INDX(a,b,c,nu)] ) * two
   	              -   v3[INDX(c,a,b,nu)] + v3[INDX(c,b,a,nu)];

        double t3_ab5 = ( v3[INDX(c,a,b,nu)] - v3[INDX(c,b,a,nu)] ) * two
	              -   v3[INDX(b,a,c,nu)] + v3[INDX(a,b,c,nu)];

        double t3_ab4 = ( v3[INDX(b,c,a,nu)] - v3[INDX(a,c,b,nu)] ) * two
	              -   v3[INDX(c,b,a,nu)] + v3[INDX(c,a,b,nu)];

        double t3_ab6 = ( v3[INDX(c,b,a,nu)] - v3[INDX(c,a,b,nu)] ) * two
	              -   v3[INDX(b,c,a,nu)] + v3[INDX(a,c,b,nu)];
#endif
#else
        double abc = fetch_x_v3( INDX(a, b, c, nu) );
        double acb = fetch_x_v3( INDX(a, c, b, nu) );
        double bac = fetch_x_v3( INDX(b, a, c, nu) );
        double bca = fetch_x_v3( INDX(b, c, a, nu) );
        double cba = fetch_x_v3( INDX(c, b, a, nu) );
        double cab = fetch_x_v3( INDX(c, a, b, nu) );

        double t3_ab1 = ( abc - bac ) * two
   	              -   acb + bca;

        double t3_ab2 = ( acb - bca ) * two
	              -   abc + bac;

        double t3_ab3 = ( bac - abc ) * two
   	              -   cab + cba;

        double t3_ab5 = ( cab - cba )  * two
	              -   bac + abc;

        double t3_ab4 = ( bca - acb ) * two
	              -   cba + cab;

        double t3_ab6 = ( cba - cab ) * two
	              -   bca + acb;
#endif

        t1bi += ( t3_ab1*voe_jk[INDX(a,c,0,nu)] 
	     +    t3_ab2*voe_kj[INDX(a,c,0,nu)] ) * denom * om;

        t1bj += ( t3_ab3*voe_ik[INDX(a,c,0,nu)] 
	     +    t3_ab5*voe_ki[INDX(a,c,0,nu)] ) * denom * om;

        t1bk += ( t3_ab4*voe_ij[INDX(a,c,0,nu)] 
	     +    t3_ab6*voe_ji[INDX(a,c,0,nu)] ) * denom * om;

      } /* end if */

loop_end1:

    } /* end idx loop */

  } /* end a loop */


  __syncthreads();


  etd_shared[ti1d] = t1bi;

  __syncthreads();

  offi = INDX(b,i-1,0,nu);
#if 0
  if( ti1d == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempi += etd_shared[idx];
    } /* end for */
    t1[offi] += tempi;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( ti1d == 0 ) t1[offi] += etd_shared[0];

  __syncthreads();

  etd_shared[ti1d] = t1bj;

  __syncthreads();

  offj = INDX(b,j-1,0,nu);
#if 0
  if( ti1d == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempj += etd_shared[idx];
    } /* end for */
    t1[offj] += tempj;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( ti1d == 0 ) t1[offj] += etd_shared[0];

  __syncthreads();

  etd_shared[ti1d] = t1bk;

  __syncthreads();

  offk = INDX(b,k-1,0,nu);
#if 0
  if( ti1d == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempk += etd_shared[idx];
    } /* end for */
    t1[offk] += tempk;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( ti1d == 0 ) t1[offk] += etd_shared[0];
#endif
} /* end t1a_cuda_kernel */



__global__ void reduce_etd_kernel( const long int size, const double *a,
     double *result )
{
  long int i;
  for( i = 0; i < size; i++ ) 
  {
    result[0] += a[i];
  } /* end for */
  return;
} /* end kernel */

//} /* end extern C */

#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "cuda_kernels.h"

#define INDX(a,b,c,ld) ( ( (c) * (ld) * (ld) ) \
                       + ( (b) * (ld) ) \
                       + ( (a) ) )

extern "C" {

__device__ void warpReduce( volatile double *sdata )
{
#if 1
  if( blockDim.x >= 64 && blockDim.x % 32 == 0 )
  {
    for( unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
      if( threadIdx.x < s ) 
         sdata[threadIdx.x] += sdata[threadIdx.x + s];
      __syncthreads();
    } /* end for */
    if( threadIdx.x < 32 ) 
    {
      sdata[threadIdx.x] += sdata[threadIdx.x + 32];
      sdata[threadIdx.x] += sdata[threadIdx.x + 16];
      sdata[threadIdx.x] += sdata[threadIdx.x +  8];
      sdata[threadIdx.x] += sdata[threadIdx.x +  4];
      sdata[threadIdx.x] += sdata[threadIdx.x +  2];
      sdata[threadIdx.x] += sdata[threadIdx.x +  1];
    } /* end if */
  }
  else
#endif
  {
    if( threadIdx.x == 0 )
    {
      for( int idx = 1; idx < blockDim.x; idx++ )
      {
        sdata[0] += sdata[idx];
      } /* end for */
    } /* end if */
  } /* end */
} /* end warp Reduce */

__global__ void etd_cuda_kernel( const int i, const int j, const int k, 
        const int no, const int nu,
        const double *v3, const double *voe_ij, const double *voe_ji, 
        const double *voe_ik, const double *voe_ki, 
        const double *voe_jk, const double *voe_kj,
        double *t1, const double *eh, const double *ep, double *etd_reduce )
{

__shared__ double etd_shared[SHARED_REDUCTION_SIZE];

  int a = blockIdx.x;
  int b = blockIdx.y;
  double dijk = eh[i-1] + eh[j-1] + eh[k-1];
  double x3 = 0.0;
  const double two = 2.0, four = 4.0, eight = 8.0, om = -1.0;
  double d1,d2,d3,f, t1ai = 0.0, t1bi = 0.0;
  double t1aj = 0.0, t1bj = 0.0, t1ak = 0.0, t1bk = 0.0;

  for( int idx = 0; idx < nu; idx += blockDim.x )
  {
    int c = idx + threadIdx.x;

/*
 * don't do the loop if my id is outside the bounds of nu
 */

    if( c < nu )
    {

      if( a > b ) goto loop_end;
      if( a == b && b == c ) goto loop_end;
      double dabc = ep[a] + ep[b] + ep[c];
      double denom = 1.0 / ( dijk - dabc );

      d1 = v3[INDX(a, b, c, nu)];
      d2 = v3[INDX(a, c, b, nu)] + v3[INDX(c, b, a, nu)] 
                + v3[INDX(b, a, c, nu)];
      d3 = v3[INDX(b, c, a, nu)] + v3[INDX(c, a, b, nu)];
      f  = d1*eight - d2*four + d3*two;
      x3        += f*d1*denom;

      if( a == b ) goto loop_end;

      d1 = v3[INDX(b, a, c, nu)];
      d2 = v3[INDX(b, c, a, nu)] + v3[INDX(c, a, b, nu)] 
         + v3[INDX(a, b, c, nu)];
      d3 = v3[INDX(a, c, b, nu)] + v3[INDX(c, b, a, nu)];
      f  = d1*eight - d2*four + d3*two;
      x3 += f*d1*denom;

    } /* end if */
    
loop_end:
  } /* end for */


  etd_shared[threadIdx.x] = 0.0;
  etd_shared[threadIdx.x] = x3;

  __syncthreads();

  int offset = INDX(a, b, 0, gridDim.x );

  double temp = 0.0;

#if 0
  if( threadIdx.x == 0 )
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      temp += etd_shared[idx];
    } /* end for */
    etd_reduce[offset] = temp;
  } /* end if */
#endif
#if 0
  for( unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
  {
    if( threadIdx.x < s ) 
       etd_shared[threadIdx.x] += etd_shared[threadIdx.x + s];
    __syncthreads();
  } /* end for */

  if( threadIdx.x < 32 ) warpReduce( etd_shared, threadIdx.x );

  if( threadIdx.x == 0 ) etd_reduce[offset] = etd_shared[0];
#endif
#if 1
  warpReduce( etd_shared );
#endif
  __syncthreads();
  if( threadIdx.x == 0 ) etd_reduce[offset] = etd_shared[0];



//  etd_shared[threadIdx.x] = t1ai;

} /* end kernel */


__global__ void t1a_cuda_kernel( const int i, const int j, const int k, 
        const int no, const int nu,
        const double *v3, const double *voe_ij, const double *voe_ji, 
        const double *voe_ik, const double *voe_ki, 
        const double *voe_jk, const double *voe_kj,
        double *t1, const double *eh, const double *ep, double *etd_reduce )
{

__shared__ double etd_shared[SHARED_REDUCTION_SIZE];

  int a = blockIdx.x;
  int b;
  double dijk = eh[i-1] + eh[j-1] + eh[k-1];
  const double two = 2.0, four = 4.0, eight = 8.0, om = -1.0;
  double t1ai = 0.0, t1aj = 0.0, t1ak = 0.0;
  int c_major, c_minor;
  __shared__ double BUFFER_cab[16][128];		// BUFFER_cab[c_minor][threadIdx.x] <=> v3[INDX(c,a,b,nu)]
  __shared__ double BUFFER_cba[16][128];		// BUFFER_cba[c_minor][threadIdx.x] <=> v3[INDX(c,b,a,nu)]

  for( int idx = 0; idx < nu; idx += blockDim.x )
  {
        b = idx + threadIdx.x;
	  int temp = threadIdx.x - (threadIdx.x%16);
      for( c_major = 0; c_major < nu; c_major += 16 ) {
         for( c_minor = 0; c_minor < 16; c_minor++ ) {
	      BUFFER_cab[threadIdx.x%16][temp + c_minor] = v3[INDX(c_major + (threadIdx.x %16),a,idx + c_minor + temp,nu)];
	      BUFFER_cba[threadIdx.x%16][temp + c_minor] = v3[INDX(c_major + (threadIdx.x %16),idx + c_minor + temp,a,nu)];
         }
		 __syncthreads();
    if( b < nu && a < b )
    {
      for( c_minor = 0; c_minor < 16; c_minor++ )
      {
	    int c = c_major + c_minor;
		if ( c >= nu ) break;						// Break to c_major loop, if syncthreads allow.
        double dabc = ep[a] + ep[b] + ep[c];
        double denom = 1.0 / ( dijk - dabc );

                double abcbac = v3[INDX(a,b,c,nu)] - v3[INDX(b,a,c,nu)];
                double acbbca = v3[INDX(a,c,b,nu)] - v3[INDX(b,c,a,nu)];
//              double cabcba = v3[INDX(c,a,b,nu)] - v3[INDX(c,b,a,nu)];
              double cabcba = BUFFER_cab[c_minor][threadIdx.x] - BUFFER_cba[c_minor][threadIdx.x];

        double t3_ab1 = abcbac * two - acbbca;
        double t3_ab2 = acbbca * two - abcbac;
        double t3_ab3 = - ( abcbac * two + cabcba );
                double t3_ab4 = ( - acbbca * two ) + cabcba;
                double t3_ab5 = cabcba * two + abcbac;
                double t3_ab6 = ( - cabcba * two ) + acbbca;
/*
        double t3_ab1 = ( v3[INDX(a,b,c,nu)] - v3[INDX(b,a,c,nu)] ) * two
                      -   v3[INDX(a,c,b,nu)] + v3[INDX(b,c,a,nu)];

        double t3_ab2 = ( v3[INDX(a,c,b,nu)] - v3[INDX(b,c,a,nu)] ) * two
                      -   v3[INDX(a,b,c,nu)] + v3[INDX(b,a,c,nu)];

        double t3_ab3 = ( v3[INDX(b,a,c,nu)] - v3[INDX(a,b,c,nu)] ) * two
                      -   v3[INDX(c,a,b,nu)] + v3[INDX(c,b,a,nu)];

        double t3_ab5 = ( v3[INDX(c,a,b,nu)] - v3[INDX(c,b,a,nu)] ) * two
                      -   v3[INDX(b,a,c,nu)] + v3[INDX(a,b,c,nu)];

        double t3_ab4 = ( v3[INDX(b,c,a,nu)] - v3[INDX(a,c,b,nu)] ) * two
                      -   v3[INDX(c,b,a,nu)] + v3[INDX(c,a,b,nu)];

        double t3_ab6 = ( v3[INDX(c,b,a,nu)] - v3[INDX(c,a,b,nu)] ) * two
                      -   v3[INDX(b,c,a,nu)] + v3[INDX(a,c,b,nu)];
*/
//      if( a == 0 && b == 4 ) printf("c %d t3_ab1 %22.17e t3_ab2 %22.17e\n",
//         c,t3_ab1,t3_ab2);

        t1ai += ( t3_ab1*voe_jk[INDX(b,c,0,nu)] 
             +    t3_ab2*voe_kj[INDX(b,c,0,nu)] ) * denom;

        t1aj += ( t3_ab3*voe_ik[INDX(b,c,0,nu)] 
             +    t3_ab5*voe_ki[INDX(b,c,0,nu)] ) * denom;

        t1ak += ( t3_ab4*voe_ij[INDX(b,c,0,nu)] 
             +    t3_ab6*voe_ji[INDX(b,c,0,nu)] ) * denom;

      } /* end c_minor loop */
    } /* end if */
      } /* end c_major loop */
  } /* end idx loop */


  etd_shared[threadIdx.x] = t1ai;

  __syncthreads();

  double tempi = 0.0, tempj = 0.0, tempk = 0.0;

  int offi = INDX(a,i-1,0,nu);
#if 0
  if( threadIdx.x == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempi += etd_shared[idx];
    } /* end for */
    t1[offi] += tempi;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( threadIdx.x == 0 ) t1[offi] += etd_shared[0];

  __syncthreads();

  etd_shared[threadIdx.x] = t1aj;

  __syncthreads();

  int offj = INDX(a,j-1,0,nu);
#if 0
  if( threadIdx.x == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempj += etd_shared[idx];
    } /* end for */
    t1[offj] += tempj;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( threadIdx.x == 0 ) t1[offj] += etd_shared[0];

  __syncthreads();

  etd_shared[threadIdx.x] = t1ak;

  __syncthreads();

  int offk = INDX(a,k-1,0,nu);
#if 0
  if( threadIdx.x == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempk += etd_shared[idx];
    } /* end for */
    t1[offk] += tempk;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( threadIdx.x == 0 ) t1[offk] += etd_shared[0];

//    Next calculation:

  double t1bi = 0.0, t1bj = 0.0, t1bk = 0.0;
  dijk = eh[i-1] + eh[j-1] + eh[k-1];

  b = blockIdx.x;
  for( int idx = 0; idx < nu; idx += blockDim.x )
  {
        a = idx + threadIdx.x;
/*
 * don't do the loop if my id is outside the bounds of nu
 */
	  int temp = threadIdx.x - (threadIdx.x%16);
      for( c_major = 0; c_major < nu; c_major += 16 ) {
         for( c_minor = 0; c_minor < 16; c_minor++ ) {
	      BUFFER_cab[threadIdx.x%16][temp + c_minor] = v3[INDX(c_major + (threadIdx.x %16),idx + c_minor + temp,b,nu)];
	      BUFFER_cba[threadIdx.x%16][temp + c_minor] = v3[INDX(c_major + (threadIdx.x %16),b,idx + c_minor + temp,nu)];
/*
int c = c_major + c_minor;
BUFFER_cab[c_minor][threadIdx.x] = v3[INDX(c,a,b,nu)];
BUFFER_cba[c_minor][threadIdx.x] = v3[INDX(c,b,a,nu)];
*/
         }
		 __syncthreads();
    if( a < nu && a < b )
    {
      for( c_minor = 0; c_minor < 16; c_minor++ )
      {
	    int c = c_major + c_minor;
		if ( c >= nu ) break;						// Break to c_major loop, if syncthreads allow.
        double dabc = ep[a] + ep[b] + ep[c];
        double denom = 1.0 / ( dijk - dabc );

                double abcbac = v3[INDX(a,b,c,nu)] - v3[INDX(b,a,c,nu)];
                double acbbca = v3[INDX(a,c,b,nu)] - v3[INDX(b,c,a,nu)];
//                double cabcba = v3[INDX(c,a,b,nu)] - v3[INDX(c,b,a,nu)];
              double cabcba = BUFFER_cab[c_minor][threadIdx.x] - BUFFER_cba[c_minor][threadIdx.x];
/*
if ( BUFFER_cab[c_minor][threadIdx.x] != v3[INDX(c,a,b,nu)] )
	printf("ThreadIdx.x=%d c_minor=%d c_major=%d c=%d a=%d b=%d nu=%d BUFFER_cab[c_minor][threadIdx.x]=%g v3[INDX(c,a,b,nu)]=%g\n",
	       threadIdx.x, c_minor, c_major, c, a, b, nu, BUFFER_cab[c_minor][threadIdx.x], v3[INDX(c,a,b,nu)] );
*/
        double t3_ab1 = abcbac * two - acbbca;
        double t3_ab2 = acbbca * two - abcbac;
        double t3_ab3 = - ( abcbac * two + cabcba );
                double t3_ab4 = ( - acbbca * two ) + cabcba;
                double t3_ab5 = cabcba * two + abcbac;
                double t3_ab6 = ( - cabcba * two ) + acbbca;
/*
        double t3_ab1 = ( v3[INDX(a,b,c,nu)] - v3[INDX(b,a,c,nu)] ) * two
                      -   v3[INDX(a,c,b,nu)] + v3[INDX(b,c,a,nu)];

        double t3_ab2 = ( v3[INDX(a,c,b,nu)] - v3[INDX(b,c,a,nu)] ) * two
                      -   v3[INDX(a,b,c,nu)] + v3[INDX(b,a,c,nu)];

        double t3_ab3 = ( v3[INDX(b,a,c,nu)] - v3[INDX(a,b,c,nu)] ) * two
                      -   v3[INDX(c,a,b,nu)] + v3[INDX(c,b,a,nu)];

        double t3_ab5 = ( v3[INDX(c,a,b,nu)] - v3[INDX(c,b,a,nu)] ) * two
                      -   v3[INDX(b,a,c,nu)] + v3[INDX(a,b,c,nu)];

        double t3_ab4 = ( v3[INDX(b,c,a,nu)] - v3[INDX(a,c,b,nu)] ) * two
                      -   v3[INDX(c,b,a,nu)] + v3[INDX(c,a,b,nu)];

        double t3_ab6 = ( v3[INDX(c,b,a,nu)] - v3[INDX(c,a,b,nu)] ) * two
                      -   v3[INDX(b,c,a,nu)] + v3[INDX(a,c,b,nu)];
*/
//      if( a == 0 && b == 4 ) printf("c %d t3_ab1 %22.17e t3_ab2 %22.17e\n",
//         c,t3_ab1,t3_ab2);

        t1bi += ( t3_ab1*voe_jk[INDX(a,c,0,nu)] 
             +    t3_ab2*voe_kj[INDX(a,c,0,nu)] ) * denom * om;

        t1bj += ( t3_ab3*voe_ik[INDX(a,c,0,nu)] 
             +    t3_ab5*voe_ki[INDX(a,c,0,nu)] ) * denom * om;

        t1bk += ( t3_ab4*voe_ij[INDX(a,c,0,nu)] 
             +    t3_ab6*voe_ji[INDX(a,c,0,nu)] ) * denom * om;

      } /* end c_minor loop */
    } /* end if */
      } /* end c_major loop */
  } /* end idx loop */


  __syncthreads();


  etd_shared[threadIdx.x] = t1bi;

  __syncthreads();

  tempi = 0.0, tempj = 0.0, tempk = 0.0;

  offi = INDX(b,i-1,0,nu);
#if 0
  if( threadIdx.x == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempi += etd_shared[idx];
    } /* end for */
    t1[offi] += tempi;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( threadIdx.x == 0 ) t1[offi] += etd_shared[0];

  __syncthreads();

  etd_shared[threadIdx.x] = t1bj;

  __syncthreads();

  offj = INDX(b,j-1,0,nu);
#if 0
  if( threadIdx.x == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempj += etd_shared[idx];
    } /* end for */
    t1[offj] += tempj;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( threadIdx.x == 0 ) t1[offj] += etd_shared[0];

  __syncthreads();

  etd_shared[threadIdx.x] = t1bk;

  __syncthreads();

  offk = INDX(b,k-1,0,nu);
#if 0
  if( threadIdx.x == 0 ) 
  {
    for( int idx = 0; idx < blockDim.x; idx++ )
    {
      tempk += etd_shared[idx];
    } /* end for */
    t1[offk] += tempk;
  } /* end if */
#endif
  warpReduce( etd_shared );
  __syncthreads();
  if( threadIdx.x == 0 ) t1[offk] += etd_shared[0];
} /* end t1a_cuda_kernel_b */



__global__ void reduce_etd_kernel( const long int size, const double *a,
     double *result )
{
  long int i;
  for( i = 0; i < size; i++ ) 
  {
    result[0] += a[i];
  } /* end for */
  return;
} /* end kernel */

} /* end extern C */

